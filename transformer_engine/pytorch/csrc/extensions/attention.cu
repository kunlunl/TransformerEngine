#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"

constexpr int block_size = 512;
constexpr int ctas_per_sm = 4;

// get the fused attention backend
NVTE_Fused_Attn_Backend get_fused_attn_backend(
                const transformer_engine::DType q_dtype,
                const transformer_engine::DType kv_dtype,
                NVTE_QKV_Layout qkv_layout,
                NVTE_Bias_Type bias_type,
                NVTE_Mask_Type attn_mask_type,
                float p_dropout,
                size_t num_attn_heads, size_t num_gqa_groups,
                size_t max_seqlen_q, size_t max_seqlen_kv,
                size_t head_dim) {
  NVTE_Fused_Attn_Backend fused_attention_backend =
          nvte_get_fused_attn_backend(
                          static_cast<NVTEDType>(q_dtype), static_cast<NVTEDType>(kv_dtype),
                          qkv_layout, bias_type, attn_mask_type, p_dropout,
                          num_attn_heads, num_gqa_groups,
                          max_seqlen_q, max_seqlen_kv, head_dim);
  return fused_attention_backend;
}

// fast zero-fills of tensors
template <typename scalar_t>
__global__ void __launch_bounds__(block_size) mha_fill_kernel(scalar_t* out_tensor,
                const int32_t* const start_row,
                const size_t num_rows) {
  size_t row_stride = gridDim.y * blockDim.x;
  size_t row_index = blockIdx.x + static_cast<size_t>(start_row[0]);
  size_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  while (row_index < num_rows) {
    out_tensor[row_index*row_stride + col_index] = 0;
    row_index += gridDim.x;
  }
}

// fast zero-fills of tensors
void mha_fill(const at::Tensor &self, const at::Tensor &start_index) {
  auto max_tokens = self.size(0);
  auto self_2d = self.view({max_tokens, -1});
  auto fcd_size = self_2d.size(1);
  TORCH_CHECK(self.is_contiguous(), "input not contiguous");
  TORCH_CHECK(fcd_size % block_size == 0, "input size not aligned to block size");
  const int num_mp = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  uint64_t num_blk_y = (uint64_t)(fcd_size / block_size);
  uint64_t num_blk_x = (uint64_t)((num_mp * ctas_per_sm + num_blk_y - 1) / num_blk_y);
  dim3 dim_grid(num_blk_x, num_blk_y);
  dim3 dim_block(block_size);
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
          at::ScalarType::Half, at::ScalarType::BFloat16,
          self_2d.scalar_type(), "mha_fill", [&]() {
          mha_fill_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                  self_2d.data_ptr<scalar_t>(),
                  static_cast<int32_t*>(start_index.data_ptr()),
                  max_tokens);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
}

// extract seed and offset from PhiloxCudaState
__global__ void unpack(at::PhiloxCudaState arg, int64_t* rng_state_ptr) {
  if (arg.captured_) {
    rng_state_ptr[0] = static_cast<int64_t>(*arg.seed_.ptr);
    rng_state_ptr[1] = static_cast<int64_t>(
                    *(arg.offset_.ptr) + static_cast<int64_t>(arg.offset_intragraph_));
  } else {
    rng_state_ptr[0] = static_cast<int64_t>(arg.seed_.val);
    rng_state_ptr[1] = static_cast<int64_t>(arg.offset_.val);
  }
}

// extract PhiloxCudaState from CUDA random number generator
at::PhiloxCudaState init_philox_state(
                at::CUDAGeneratorImpl* gen,
                size_t elts_per_thread) {
  at::PhiloxCudaState philox_args;
  std::lock_guard<std::mutex> lock(gen->mutex_);
  philox_args = gen->philox_cuda_state(elts_per_thread);
  return philox_args;
}

// fused attention FWD with packed QKV
std::vector<at::Tensor> fused_attn_fwd_qkvpacked(
                size_t max_seqlen, bool is_training, float attn_scale,
                float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto qkv_sizes = QKV.sizes().vec();
  std::vector<size_t> qkv_shape{qkv_sizes.begin(), qkv_sizes.end()};
  std::vector<size_t> q_shape;
  for (auto i : qkv_shape) {
    if (i != 3) {
      q_shape.push_back(i);
    }
  }
  std::vector<int64_t> o_shape{q_shape.begin(), q_shape.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty(o_shape, options);

  // construct NVTE tensors
  TensorWrapper te_QKV, te_S, te_O, te_Bias, te_cu_seqlens;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_sizes = cu_seqlens.sizes().vec();
  std::vector<size_t> cu_seqlens_shape{cu_seqlens_sizes.begin(), cu_seqlens_sizes.end()};
  te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), cu_seqlens_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract random number generator seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed QKV
std::vector<at::Tensor> fused_attn_bwd_qkvpacked(
                size_t max_seqlen, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto qkv_sizes = QKV.sizes().vec();
  std::vector<size_t> qkv_shape{qkv_sizes.begin(), qkv_sizes.end()};
  std::vector<size_t> q_shape;
  for (auto i : qkv_shape) {
    if (i != 3) {
      q_shape.push_back(i);
    }
  }
  auto h = q_shape[q_shape.size() - 2];

  // create output tensor dQKV
  at::Tensor dQKV = torch::empty_like(QKV);
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);

  // construct NVTE tensors
  TensorWrapper te_QKV, te_O, te_dO, te_S, te_dP, te_dQKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(dQKV, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQKV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    nullptr, scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), qkv_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // convert auxiliary tensors from forward into NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS)
    && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h),
                    static_cast<int64_t>(max_seqlen),
                    static_cast<int64_t>(max_seqlen)}, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_sizes = cu_seqlens.sizes().vec();
  std::vector<size_t> cu_seqlens_shape{cu_seqlens_sizes.begin(), cu_seqlens_sizes.end()};
  TensorWrapper te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), cu_seqlens_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQKV, dBias};
}

// fused attention FWD with packed KV
std::vector<at::Tensor> fused_attn_fwd_kvpacked(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto kv_sizes = KV.sizes().vec();
  std::vector<size_t> kv_shape{kv_sizes.begin(), kv_sizes.end()};
  std::vector<int64_t> o_shape{q_shape.begin(), q_shape.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty(o_shape, options);

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_S, te_O, te_Bias, te_cu_seqlens_q, te_cu_seqlens_kv;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed KV
std::vector<at::Tensor> fused_attn_bwd_kvpacked(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto kv_sizes = KV.sizes().vec();
  std::vector<size_t> kv_shape{kv_sizes.begin(), kv_sizes.end()};
  std::vector<size_t> k_shape;
  for (auto i : kv_shape) {
    if (i != 2) {
      k_shape.push_back(i);
    }
  }
  auto h_q = q_shape[q_shape.size() - 2];
  auto h_kv = k_shape[k_shape.size() - 2];
  auto d = q_shape[q_shape.size() - 1];

  // create output tensors dQ and dKV
  at::Tensor dQ = torch::empty_like(Q);
  at::Tensor dKV = torch::empty_like(KV);
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_O, te_dO, te_S, te_dP, te_dQ, te_dKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if (set_zero && ((h_q * d)% block_size == 0) && ((h_kv * d)% block_size == 0)) {
      mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dKV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dKV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                    scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), kv_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS)
    && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h_q),
                    static_cast<int64_t>(max_seqlen_q),
                    static_cast<int64_t>(max_seqlen_kv)}, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dKV, dBias};
}

// fused attention FWD with separate Q, K and V tensors
std::vector<at::Tensor> fused_attn_fwd(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor K,
                const at::Tensor V,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto k_sizes = K.sizes().vec();
  std::vector<size_t> k_shape{k_sizes.begin(), k_sizes.end()};
  auto v_sizes = V.sizes().vec();
  std::vector<size_t> v_shape{v_sizes.begin(), v_sizes.end()};

  // create output tensor O
  auto O = torch::empty_like(Q);

  // construct NVTE tensors
  TensorWrapper te_Q, te_K, te_V, te_S, te_O, te_Bias;
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto options = torch::TensorOptions().dtype(torch::kInt64).device(torch::kCUDA);
  auto rng_state = torch::empty({2}, options);
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with separate Q, K and V
std::vector<at::Tensor> fused_attn_bwd(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor K,
                const at::Tensor V,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto k_sizes = K.sizes().vec();
  std::vector<size_t> k_shape{k_sizes.begin(), k_sizes.end()};
  auto v_sizes = V.sizes().vec();
  std::vector<size_t> v_shape{v_sizes.begin(), v_sizes.end()};
  auto h_q = q_shape[q_shape.size() - 2];
  auto h_kv = k_shape[k_shape.size() - 2];
  auto d = q_shape[q_shape.size() - 1];
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);

  at::Tensor dQ;
  at::Tensor dK;
  at::Tensor dV;
  at::Tensor dQKV, dKV;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  std::vector<int64_t> tmp_shape;
  switch (layout_group) {
      case NVTE_QKV_Layout_Group::NVTE_3HD:
          tmp_shape = std::vector<int64_t>{q_sizes.begin(), q_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(3));
          dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          break;
      case NVTE_QKV_Layout_Group::NVTE_H3D:
          tmp_shape = std::vector<int64_t>{q_sizes.begin(), q_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(3));
          dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_2HD:
          dQ = torch::empty_like(Q);
          tmp_shape = std::vector<int64_t>{k_sizes.begin(), k_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(2));
          dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_H2D:
          dQ = torch::empty_like(Q);
          tmp_shape = std::vector<int64_t>{k_sizes.begin(), k_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(2));
          dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_HD_HD:
          dQ = torch::empty_like(Q);
          dK = torch::empty_like(K);
          dV = torch::empty_like(V);
          break;
      default:
          NVTE_ERROR("QKV layout not supported!");
    }

  // construct NVTE tensors
  TensorWrapper te_Q, te_K, te_V, te_O, te_dO, te_S, te_dP, te_dQ, te_dK, te_dV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if (set_zero
          && ((h_q * d) % block_size == 0)
          && ((h_kv * d) % block_size == 0)
          && dQ.is_contiguous()
          && dK.is_contiguous()
          && dV.is_contiguous()) {
      mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dK, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dK.fill_(0);
      dV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                    scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dK = makeTransformerEngineTensor(dK.data_ptr(), k_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dV = makeTransformerEngineTensor(dV.data_ptr(), v_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dK = makeTransformerEngineTensor(dK.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dV = makeTransformerEngineTensor(dV.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS)
    && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h_q),
                    static_cast<int64_t>(max_seqlen_q),
                    static_cast<int64_t>(max_seqlen_kv)}, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dK.data(),
                  te_dV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dK.data(),
                  te_dV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dK, dV, dBias};
}

namespace flash_attention {

constexpr int warp_size = 32;
constexpr int type_size = 2;  // FP16 or BF16
constexpr int nvec = sizeof(uint64_t) / type_size;
constexpr int load_size = warp_size * nvec;
constexpr int block_size = 512;

template <typename T>
__launch_bounds__(block_size)
__global__ void prepare_kernel_fwd(const T *qkvi,
                                   T *qkv,
                                   const size_t B,
                                   const size_t S,
                                   const size_t Z,
                                   const size_t W) {
    const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
    const int id_in_warp = threadIdx.x % warp_size;
    const size_t offset_input = blockIdx.y * W + warpid * 3 * W * Z + id_in_warp * nvec;
    const T *my_input = qkvi + offset_input;

    const size_t s = warpid / B;
    if (s >= S) return;

    const size_t b = warpid % B;

    const size_t offset_output = blockIdx.y * B * S * Z * W +
                                 (s + b * S) * W * Z +
                                 id_in_warp * nvec;

    T *my_output = qkv + offset_output;

    for (int i = 0; i < Z; ++i) {
        uint64_t *out = reinterpret_cast<uint64_t*>(my_output + i * load_size);
        *out = *reinterpret_cast<const uint64_t*>(my_input + i * load_size * 3);
    }
}

template <typename T>
__launch_bounds__(block_size)
__global__ void prepare_kernel_bwd(const T *q, const T *k, const T *v,
                                   T *qkv, const size_t B, const size_t S,
                                   const size_t Z, const size_t W) {
    const T *input = blockIdx.y == 0 ? q : (blockIdx.y == 1 ? k : v);

    const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
    const int id_in_warp = threadIdx.x % warp_size;
    const size_t offset_input = warpid * W * Z + id_in_warp * nvec;
    const T *my_input = input + offset_input;

    const size_t b = warpid / S;
    if (b >= B) return;

    const size_t s = warpid % S;

    const size_t offset_output = (b + s * B) * 3 * W * Z +
                                 id_in_warp * nvec + blockIdx.y * W;

    T *my_output = qkv + offset_output;

    for (int i = 0; i < Z; ++i) {
        uint64_t *out = reinterpret_cast<uint64_t*>(my_output + i * load_size * 3);
        *out = *reinterpret_cast<const uint64_t*>(my_input + i * load_size);
    }
}

}  // namespace flash_attention

at::Tensor fa_prepare_fwd(at::Tensor qkvi) {
    NVTE_CHECK(qkvi.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(qkvi.scalar_type() == at::ScalarType::Half ||
               qkvi.scalar_type() == at::ScalarType::BFloat16);
    NVTE_CHECK(qkvi.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(qkvi.size(3) == flash_attention::load_size);
    NVTE_CHECK(qkvi.stride(3) == 1, "Wrong stride.");
    NVTE_CHECK(qkvi.stride(2) == 3 * qkvi.size(3), "Wrong stride.");
    NVTE_CHECK(qkvi.stride(1) == 3 * qkvi.size(3) * qkvi.size(2), "Wrong stride.");
    NVTE_CHECK(qkvi.stride(0) == 3 * qkvi.size(3) * qkvi.size(2) * qkvi.size(1), "Wrong stride.");

    // [s, b, n, h * 3] -> [3, b, s, n, h]
    std::vector<int64_t> shape = {3, qkvi.size(1), qkvi.size(0), qkvi.size(2), qkvi.size(3)};
    at::Tensor qkv = at::empty(shape, at::CUDA(qkvi.scalar_type()));

    size_t warps = qkvi.size(0) * qkvi.size(1);
    size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
    size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
    dim3 grid(blocks, 3);
    int threads = flash_attention::block_size;
    if (qkvi.scalar_type() == at::ScalarType::Half) {
        using dtype = at::Half;
        flash_attention::prepare_kernel_fwd<dtype><<<grid, threads, 0,
                                                     at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            shape[1],
            shape[2],
            shape[3],
            shape[4]);
    } else {
        using dtype = at::BFloat16;
        flash_attention::prepare_kernel_fwd<dtype><<<grid, threads, 0,
                                                     at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            shape[1],
            shape[2],
            shape[3],
            shape[4]);
    }

    return qkv;
}

at::Tensor fa_prepare_bwd(at::Tensor q, at::Tensor k, at::Tensor v) {
    NVTE_CHECK(q.is_contiguous());
    NVTE_CHECK(k.is_contiguous());
    NVTE_CHECK(v.is_contiguous());
    NVTE_CHECK(q.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(k.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(v.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(q.scalar_type() == at::ScalarType::Half ||
               q.scalar_type() == at::ScalarType::BFloat16);
    NVTE_CHECK(k.scalar_type() == q.scalar_type());
    NVTE_CHECK(v.scalar_type() == q.scalar_type());
    NVTE_CHECK(q.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(q.size(3) == flash_attention::load_size);
    NVTE_CHECK(k.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(k.size(3) == flash_attention::load_size);
    NVTE_CHECK(v.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(v.size(3) == flash_attention::load_size);

    // 3 x [s, b, n, h] -> [b, s, n, 3 * h]

    std::vector<int64_t> shape = {q.size(1), q.size(0), q.size(2), 3 * q.size(3)};
    at::Tensor qkv = at::empty(shape, at::CUDA(q.scalar_type()));

    size_t warps = q.size(0) * q.size(1);
    size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
    size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
    dim3 grid(blocks, 3);
    int threads = flash_attention::block_size;
    if (q.scalar_type() == at::ScalarType::Half) {
        using dtype = at::Half;
        flash_attention::prepare_kernel_bwd<dtype><<<grid, threads, 0,
                                                 at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(),
            k.data_ptr<dtype>(),
            v.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            q.size(0),
            q.size(1),
            q.size(2),
            q.size(3));
    } else {
        using dtype = at::BFloat16;
        flash_attention::prepare_kernel_bwd<dtype><<<grid, threads, 0,
                                                 at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(),
            k.data_ptr<dtype>(),
            v.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            q.size(0),
            q.size(1),
            q.size(2),
            q.size(3));
    }

    return qkv;
}

/***************************************************************************************************
 * Binary search for supporting THD format in Context Parallel
 **************************************************************************************************/

__forceinline__
__device__ int binary_search(int target, int *array, int len) {
  int left = 1, right = len - 1;
  while (left < right) {
    int mid = (left + right) / 2;
    if (array[mid] <= target) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }
  return left - 1;
}

/***************************************************************************************************
 * Softmax LSE related operations
 **************************************************************************************************/

template <typename lse_dtype, typename Functor>
__global__ void thd_lse_kernel(lse_dtype *lse, float *half_lse, int *cu_seqlens,
                               int batch, int num_heads, int max_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  int num_total_tokens = cu_seqlens_s[batch];

  for (int token_id = tid; token_id < num_total_tokens; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      int row = seq_id * num_heads + head_id;
      int col = token_id - cu_seqlens_s[seq_id];
      int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];

      size_t idx = (size_t)row * max_seqlen + col + seq_len;
      size_t half_idx = (size_t)row * max_seqlen / 2 + col;

      Functor::run(lse, half_lse, idx, half_idx);
    }
  }
}

struct LseCorrectionFunctor {
  __forceinline__
  __device__ static void run(double *lse, float *half_lse, size_t idx, size_t half_idx) {
    double val = lse[idx];
    float val_per_step = half_lse[half_idx];
    double max_scale = max(val, val_per_step);
    double min_scale = min(val, val_per_step);
    lse[idx] = max_scale + log(1.0 + exp(min_scale - max_scale));
  }
};

void thd_lse_correction(at::Tensor &lse,
                        const at::Tensor &lse_per_step,
                        const at::Tensor &cu_seqlens,
                        int total_tokens) {
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Double);
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);

  NVTE_CHECK(lse.dim() == 3);
  NVTE_CHECK(lse_per_step.dim() == 3);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch      = lse.size(0);
  int num_heads  = lse.size(1);
  int max_seqlen = lse.size(2);

  NVTE_CHECK(lse_per_step.size(0) == batch);
  NVTE_CHECK(lse_per_step.size(1) == num_heads);
  NVTE_CHECK(lse_per_step.size(2) == max_seqlen / 2);
  NVTE_CHECK(cu_seqlens.size(0) == batch + 1);

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};
  thd_lse_kernel<double, LseCorrectionFunctor><<<grid, block, sizeof(int) * (batch+1),
                                                 at::cuda::getCurrentCUDAStream()>>>(
    lse.data_ptr<double>(),
    lse_per_step.data_ptr<float>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    num_heads,
    max_seqlen);
}

struct GetLseFunctor {
  __forceinline__
  __device__ static void run(float *lse, float *half_lse, size_t idx, size_t half_idx) {
    half_lse[half_idx] = lse[idx];
  }
};

at::Tensor thd_get_half_lse(const at::Tensor &lse,
                            const at::Tensor &cu_seqlens,
                            int total_tokens) {
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(lse.dim() == 3);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch      = lse.size(0);
  int num_heads  = lse.size(1);
  int max_seqlen = lse.size(2);

  NVTE_CHECK(cu_seqlens.size(0) == batch + 1);

  std::vector<int64_t> shape = {batch, num_heads, max_seqlen / 2};
  at::Tensor half_lse = at::zeros(shape, at::CUDA(lse.scalar_type()));

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};
  thd_lse_kernel<float, GetLseFunctor><<<grid, block, sizeof(int) * (batch+1),
                                         at::cuda::getCurrentCUDAStream()>>>(
    lse.data_ptr<float>(),
    half_lse.data_ptr<float>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    num_heads,
    max_seqlen);

  return half_lse;
}

/***************************************************************************************************
 * Out correction
 **************************************************************************************************/

template <typename dtype, int is_half, int tile_size>
__global__ void thd_out_correction_kernel(dtype *out,
                                          dtype *out_per_step,
                                          float *lse,
                                          float *lse_per_step,
                                          int *cu_seqlens,
                                          int batch,
                                          int num_heads,
                                          int dim_per_head,
                                          int max_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / (is_half + 1);
  }
  __syncthreads();

  int tile_id = (blockIdx.x * blockDim.x + threadIdx.x) / tile_size;
  int lane_id = threadIdx.x % tile_size;
  int num_tiles = (blockDim.x * gridDim.x) / tile_size;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_loops_per_head = dim_per_head * sizeof(dtype) / sizeof(float4);

  for (int token_id = tile_id; token_id < num_total_tokens; token_id += num_tiles) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      size_t idx, idx_per_step;

      int row = seq_id * num_heads + head_id;
      int col = token_id - cu_seqlens_s[seq_id];
      int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
      idx = (size_t)row * max_seqlen + col + seq_len * is_half;
      idx_per_step = (size_t)row * max_seqlen / (is_half + 1) + col;
      float lse_corrected_exp = exp(lse_per_step[idx_per_step] - lse[idx]);

      idx = (size_t)token_id + cu_seqlens_s[seq_id + 1] * is_half;
      idx = (idx * num_heads + head_id) * dim_per_head;
      idx_per_step = ((size_t)token_id * num_heads + head_id) * dim_per_head;
      dtype *cur_out = out + idx;
      dtype *cur_out_per_step = out_per_step + idx_per_step;

      for (int j = lane_id; j < num_loops_per_head; j += tile_size) {
        float4 data_per_step = ((float4*)cur_out_per_step)[j];
        float4 data          = ((float4*)cur_out)[j];
        dtype *p_per_step = (dtype*)&data_per_step;
        dtype *p          = (dtype*)&data;
        for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
          p[i] += p_per_step[i] * lse_corrected_exp;
        }
        ((float4*)cur_out)[j] = data;
      }
    }
  }
}

template<typename dtype, int is_half>
void thd_out_correction_helper(at::Tensor &out,
                               const at::Tensor &out_per_step,
                               const at::Tensor &lse,
                               const at::Tensor &lse_per_step,
                               const at::Tensor &cu_seqlens) {
  NVTE_CHECK(out.scalar_type() == out_per_step.scalar_type());
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);

  int total_tokens = out.size(0);
  int num_heads    = out.size(1);
  int dim_per_head = out.size(2);
  int batch      = lse.size(0);
  int max_seqlen = lse.size(2);

  NVTE_CHECK(out_per_step.size(0) == total_tokens / (is_half + 1));
  NVTE_CHECK(out_per_step.size(1) == num_heads);
  NVTE_CHECK(out_per_step.size(2) == dim_per_head);
  NVTE_CHECK(lse.size(1) == num_heads);
  NVTE_CHECK(lse_per_step.size(0) == batch);
  NVTE_CHECK(lse_per_step.size(1) == num_heads);
  NVTE_CHECK(lse_per_step.size(2) == max_seqlen / (is_half + 1));
  NVTE_CHECK(cu_seqlens.size(0) == batch + 1);

  constexpr int tile = 16;
  constexpr int block = 512;
  unsigned int grid_x = min((total_tokens / (is_half + 1) * tile + block - 1) / block, 256);
  dim3 grid = {grid_x, (unsigned int)num_heads};

  thd_out_correction_kernel<dtype, is_half, tile><<<grid, block, sizeof(int) * (batch+1),
                                                    at::cuda::getCurrentCUDAStream()>>>(
    out.data_ptr<dtype>(),
    out_per_step.data_ptr<dtype>(),
    lse.data_ptr<float>(),
    lse_per_step.data_ptr<float>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    num_heads,
    dim_per_head,
    max_seqlen);
}

void thd_out_correction(at::Tensor &out,
                        const at::Tensor &out_per_step,
                        const at::Tensor &lse,
                        const at::Tensor &lse_per_step,
                        const at::Tensor &cu_seqlens) {
  if (out.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens);
  } else if (out.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens);
  } else if (out.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens);
  } else {
    NVTE_ERROR("Unsupported dtype of out\n");
  }
}

void thd_out_correction_half(at::Tensor &out,
                             const at::Tensor &out_per_step,
                             const at::Tensor &lse,
                             const at::Tensor &lse_per_step,
                             const at::Tensor &cu_seqlens) {
  if (out.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens);
  } else if (out.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens);
  } else if (out.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens);
  } else {
    NVTE_ERROR("Unsupported dtype of out\n");
  }
}

/***************************************************************************************************
 * Read half of the q, dq, kv, dkv with THD format
 **************************************************************************************************/

__global__ void thd_get_half_tensor_kernel(void *half, void *tensor, int *cu_seqlens,
                                           int batch, int hidden_size_in_bytes, int half_idx) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int warpid = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  int laneid = threadIdx.x % 32;
  int num_warps = (blockDim.x * gridDim.x) / 32;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_float4s_per_token = hidden_size_in_bytes / sizeof(float4);

  size_t offset = num_total_tokens * (size_t)hidden_size_in_bytes;
  half = (void*)((char*)half + offset * blockIdx.y);
  tensor = (void*)((char*)tensor + 2 * offset * blockIdx.y);

  for (int token_id = warpid; token_id < num_total_tokens; token_id += num_warps) {
    int seqid = binary_search(token_id, cu_seqlens_s, batch + 1);

    size_t offset_in_bytes = (size_t)token_id * hidden_size_in_bytes;
    float4* cur_half_token = (float4*)((char*)half + offset_in_bytes);

    offset_in_bytes = ((size_t)token_id + cu_seqlens_s[seqid + half_idx]) * hidden_size_in_bytes;
    float4* cur_token = (float4*)((char*)tensor + offset_in_bytes);

    for (int idx = laneid; idx < num_float4s_per_token; idx += 32) {
      cur_half_token[idx] = cur_token[idx];
    }
  }
}

at::Tensor thd_get_half_tensor(const at::Tensor &tensor,
                               const at::Tensor &cu_seqlens,
                               int seq_dim,
                               int half_idx) {
  NVTE_CHECK(tensor.dim() == (3 + seq_dim));
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);

  int batch = cu_seqlens.size(0) - 1;
  int num_heads    = tensor.size(seq_dim + 1);
  int dim_per_head = tensor.size(seq_dim + 2);
  size_t hidden_size_in_bytes = num_heads * dim_per_head * c10::elementSize(tensor.scalar_type());

  NVTE_CHECK(hidden_size_in_bytes % 16 == 0);

  // Generate output
  std::vector<int64_t> shape(tensor.dim());
  for (size_t i = 0; i < shape.size(); i++) {
    shape[i] = tensor.size(i);
  }
  shape[seq_dim] /= 2;
  at::Tensor half = at::empty(shape, at::CUDA(tensor.scalar_type()));

  // Launch Kernel
  constexpr unsigned int block = 256;
  unsigned int grid_x = (tensor.size(seq_dim) / 2 * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= tensor.size(i);
  }
  dim3 grid = {grid_x, grid_y};
  thd_get_half_tensor_kernel<<<grid, block, sizeof(int) * (batch+1),
                               at::cuda::getCurrentCUDAStream()>>>(
    half.data_ptr(),
    tensor.data_ptr(),
    cu_seqlens.data_ptr<int>(),
    batch,
    hidden_size_in_bytes,
    half_idx);

  return half;
}

/***************************************************************************************************
 * Elementwise operations for half of the dq and half of the dkv with THD format
 **************************************************************************************************/

template <typename dtype, typename Functor, int group_size>
__global__ void thd_elementwise_half_kernel(dtype *whole, dtype *half, int *cu_seqlens,
                                            int batch, int hidden_size, int half_idx) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int group_id = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int lane_id = threadIdx.x % group_size;
  int num_groups = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[batch];

  size_t offset = num_total_tokens * (size_t)hidden_size;
  half = half + offset * blockIdx.y;
  whole = whole + 2 * offset * blockIdx.y;

  int num_inner_loops = hidden_size * sizeof(dtype) / sizeof(float4);

  for (int token_id = group_id; token_id < num_total_tokens; token_id += num_groups) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    dtype *cur_half_token = half + token_id * (size_t)hidden_size;
    dtype *cur_token = whole + (token_id + cu_seqlens_s[seq_id + half_idx]) * (size_t)hidden_size;
    for (int idx = lane_id; idx < num_inner_loops; idx += group_size) {
      Functor::run(cur_token, cur_half_token, idx);
    }
  }
}

template <typename dtype, typename Functor>
void thd_elementwise_half_helper(at::Tensor &whole,
                                 const at::Tensor &half,
                                 const at::Tensor &cu_seqlens,
                                 int seq_dim,
                                 int half_idx) {
  NVTE_CHECK(whole.dim() == (seq_dim + 3));
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int total_tokens = whole.size(seq_dim);
  int num_heads    = whole.size(seq_dim + 1);
  int dim_per_head = whole.size(seq_dim + 2);
  int batch = cu_seqlens.size(0) - 1;

  NVTE_CHECK(half.size(seq_dim) == total_tokens / 2);
  NVTE_CHECK(half.size(seq_dim + 1) == num_heads);
  NVTE_CHECK(half.size(seq_dim + 2) == dim_per_head);

  size_t hidden_size = num_heads * dim_per_head;
  NVTE_CHECK((hidden_size * c10::elementSize(whole.scalar_type())) % 16 == 0);

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= whole.size(i);
  }
  dim3 grid = {grid_x, grid_y};
  thd_elementwise_half_kernel<dtype, Functor, 32><<<grid, block, sizeof(int) * (batch+1),
                                                    at::cuda::getCurrentCUDAStream()>>>(
    whole.data_ptr<dtype>(),
    half.data_ptr<dtype>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    hidden_size,
    half_idx);
}

struct CopyFunctor {
  __forceinline__
  __device__ static void run(void *token, void *half_token, int idx) {
    ((float4*)token)[idx] = ((float4*)half_token)[idx];
  }
};

void thd_copy_half(at::Tensor &whole,
                   const at::Tensor &half,
                   const at::Tensor &cu_seqlens,
                   int seq_dim,
                   int half_idx) {
  if (whole.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    thd_elementwise_half_helper<dtype, CopyFunctor>(whole, half, cu_seqlens, seq_dim, half_idx);
  } else if (whole.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    thd_elementwise_half_helper<dtype, CopyFunctor>(whole, half, cu_seqlens, seq_dim, half_idx);
  } else if (whole.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    thd_elementwise_half_helper<dtype, CopyFunctor>(whole, half, cu_seqlens, seq_dim, half_idx);
  } else {
    NVTE_ERROR("Unsupported dtype of whole\n");
  }
}

template <typename dtype>
struct AddFunctor {
  __forceinline__
  __device__ static void run(dtype *token, dtype *half_token, int idx) {
    float4 data_half = ((float4*)half_token)[idx];
    float4 data      = ((float4*)token)[idx];

    dtype *p_half = (dtype*)(&data_half);
    dtype *p      = (dtype*)(&data);

    for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
      p[i] += p_half[i];
    }
    ((float4*)token)[idx] = data;
  }
};

void thd_add_half(at::Tensor &whole,
                  const at::Tensor &half,
                  const at::Tensor &cu_seqlens,
                  int seq_dim,
                  int half_idx) {
  if (whole.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    using Functor = AddFunctor<at::Half>;
    thd_elementwise_half_helper<dtype, Functor>(whole, half, cu_seqlens, seq_dim, half_idx);
  } else if (whole.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    using Functor = AddFunctor<at::BFloat16>;
    thd_elementwise_half_helper<dtype, Functor>(whole, half, cu_seqlens, seq_dim, half_idx);
  } else if (whole.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    using Functor = AddFunctor<float>;
    thd_elementwise_half_helper<dtype, Functor>(whole, half, cu_seqlens, seq_dim, half_idx);
  } else {
    NVTE_ERROR("Unsupported dtype of whole\n");
  }
}

/***************************************************************************************************
 * Elementwise operations for dq and dkv with THD format
 **************************************************************************************************/

template <typename dtype, typename Functor_0, typename Functor_1, int group_size>
__global__ void thd_elementwise_kernel(dtype *t1, dtype *t2, int *cu_seqlens,
                                       int batch, int hidden_size) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i];
  }
  __syncthreads();

  int group_id = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int lane_id = threadIdx.x % group_size;
  int num_groups = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[batch];

  size_t offset = num_total_tokens * (size_t)hidden_size;
  t2 = t2 + offset * blockIdx.y;
  t1 = t1 + offset * blockIdx.y;

  int num_inner_loops = hidden_size * sizeof(dtype) / sizeof(float4);

  for (int token_id = group_id; token_id < num_total_tokens; token_id += num_groups) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    int len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
    int half_idx = (token_id - cu_seqlens[seq_id]) < (len / 2) ? 0 : 1;

    dtype *cur_half_token = t2 + token_id * (size_t)hidden_size;
    dtype *cur_token = t1 + token_id * (size_t)hidden_size;
    for (int idx = lane_id; idx < num_inner_loops; idx += group_size) {
      if (half_idx == 0) {
        Functor_0::run(cur_token, cur_half_token, idx);
      } else {
        Functor_1::run(cur_token, cur_half_token, idx);
      }
    }
  }
}

template <typename dtype, typename Functor_0, typename Functor_1>
void thd_elementwise_helper(at::Tensor &t1,
                            const at::Tensor &t2,
                            const at::Tensor &cu_seqlens,
                            int seq_dim) {
  NVTE_CHECK(t1.dim() == (seq_dim + 3));
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int total_tokens = t1.size(seq_dim);
  int num_heads    = t1.size(seq_dim + 1);
  int dim_per_head = t1.size(seq_dim + 2);
  int batch = cu_seqlens.size(0) - 1;

  NVTE_CHECK(t2.size(seq_dim) == total_tokens);
  NVTE_CHECK(t2.size(seq_dim + 1) == num_heads);
  NVTE_CHECK(t2.size(seq_dim + 2) == dim_per_head);

  size_t hidden_size = num_heads * dim_per_head;
  NVTE_CHECK((hidden_size * c10::elementSize(t1.scalar_type())) % 16 == 0);

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= t1.size(i);
  }
  dim3 grid = {grid_x, grid_y};

  thd_elementwise_kernel<dtype, Functor_0, Functor_1, 32><<<grid, block, sizeof(int) * (batch+1),
                                                            at::cuda::getCurrentCUDAStream()>>>(
    t1.data_ptr<dtype>(),
    t2.data_ptr<dtype>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    hidden_size);
}

void thd_copy_add(at::Tensor &t1,
                  const at::Tensor &t2,
                  const at::Tensor &cu_seqlens,
                  int seq_dim) {
  if (t1.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    using Functor_0 = CopyFunctor;
    using Functor_1 = AddFunctor<at::Half>;
    thd_elementwise_helper<dtype, Functor_0, Functor_1>(t1, t2, cu_seqlens, seq_dim);
  } else if (t1.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    using Functor_0 = CopyFunctor;
    using Functor_1 = AddFunctor<at::BFloat16>;
    thd_elementwise_helper<dtype, Functor_0, Functor_1>(t1, t2, cu_seqlens, seq_dim);
  } else if (t1.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    using Functor_0 = CopyFunctor;
    using Functor_1 = AddFunctor<float>;
    thd_elementwise_helper<dtype, Functor_0, Functor_1>(t1, t2, cu_seqlens, seq_dim);
  } else {
    NVTE_ERROR("Unsupported dtype of t1\n");
  }
}

void thd_add_copy(at::Tensor &t1,
                  const at::Tensor &t2,
                  const at::Tensor &cu_seqlens,
                  int seq_dim) {
  if (t1.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    using Functor_0 = AddFunctor<at::Half>;
    using Functor_1 = CopyFunctor;
    thd_elementwise_helper<dtype, Functor_0, Functor_1>(t1, t2, cu_seqlens, seq_dim);
  } else if (t1.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    using Functor_0 = AddFunctor<at::BFloat16>;
    using Functor_1 = CopyFunctor;
    thd_elementwise_helper<dtype, Functor_0, Functor_1>(t1, t2, cu_seqlens, seq_dim);
  } else if (t1.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    using Functor_0 = AddFunctor<float>;
    using Functor_1 = CopyFunctor;
    thd_elementwise_helper<dtype, Functor_0, Functor_1>(t1, t2, cu_seqlens, seq_dim);
  } else {
    NVTE_ERROR("Unsupported dtype of t1\n");
  }
}

__global__ void thd_partition_indices_kernel(int *output,
                                             int *cu_seqlens,
                                             int batch,
                                             int total_tokens,
                                             int world_size,
                                             int rank) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / world_size;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (int token_id = tid; token_id < total_tokens / world_size; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
    int index = token_id - cu_seqlens_s[seq_id];
    int offset = index < seq_len/2 ? rank : (world_size-1) * 2 - rank;
    index += cu_seqlens_s[seq_id] * world_size + seq_len / 2 * offset;
    output[token_id] = index;
  }
}

at::Tensor thd_get_partitioned_indices(const at::Tensor &cu_seqlens,
                                       int total_tokens,
                                       int world_size,
                                       int rank) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);
  NVTE_CHECK(rank >= 0 && rank < world_size);
  NVTE_CHECK(world_size > 0);
  NVTE_CHECK(total_tokens > 0 && total_tokens % (world_size * 2) == 0);

  int batch = cu_seqlens.size(0) - 1;

  std::vector<int64_t> shape = {total_tokens / world_size};
  at::Tensor output = at::empty(shape, at::CUDA(at::ScalarType::Int));

  constexpr unsigned int block = 256;
  unsigned int grid = (output.size(0) + block - 1) / block;
  thd_partition_indices_kernel<<<grid, block, sizeof(int) * (batch+1),
                                 at::cuda::getCurrentCUDAStream()>>>(
    output.data_ptr<int>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    total_tokens,
    world_size,
    rank);

  return output;
}

#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"

constexpr int block_size = 512;
constexpr int ctas_per_sm = 4;

// get the fused attention backend
NVTE_Fused_Attn_Backend get_fused_attn_backend(
                const transformer_engine::DType q_dtype,
                const transformer_engine::DType kv_dtype,
                NVTE_QKV_Layout qkv_layout,
                NVTE_Bias_Type bias_type,
                NVTE_Mask_Type attn_mask_type,
                float p_dropout,
                size_t num_attn_heads, size_t num_gqa_groups,
                size_t max_seqlen_q, size_t max_seqlen_kv,
                size_t head_dim) {
  NVTE_Fused_Attn_Backend fused_attention_backend =
          nvte_get_fused_attn_backend(
                          static_cast<NVTEDType>(q_dtype), static_cast<NVTEDType>(kv_dtype),
                          qkv_layout, bias_type, attn_mask_type, p_dropout,
                          num_attn_heads, num_gqa_groups,
                          max_seqlen_q, max_seqlen_kv, head_dim);
  return fused_attention_backend;
}

// fast zero-fills of tensors
template <typename scalar_t>
__global__ void __launch_bounds__(block_size) mha_fill_kernel(scalar_t* out_tensor,
                const int32_t* const start_row,
                const size_t num_rows) {
  size_t row_stride = gridDim.y * blockDim.x;
  size_t row_index = blockIdx.x + static_cast<size_t>(start_row[0]);
  size_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  while (row_index < num_rows) {
    out_tensor[row_index*row_stride + col_index] = 0;
    row_index += gridDim.x;
  }
}

// fast zero-fills of tensors
void mha_fill(const at::Tensor &self, const at::Tensor &start_index) {
  auto max_tokens = self.size(0);
  auto self_2d = self.view({max_tokens, -1});
  auto fcd_size = self_2d.size(1);
  TORCH_CHECK(self.is_contiguous(), "input not contiguous");
  TORCH_CHECK(fcd_size % block_size == 0, "input size not aligned to block size");
  const int num_mp = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  uint64_t num_blk_y = (uint64_t)(fcd_size / block_size);
  uint64_t num_blk_x = (uint64_t)((num_mp * ctas_per_sm + num_blk_y - 1) / num_blk_y);
  dim3 dim_grid(num_blk_x, num_blk_y);
  dim3 dim_block(block_size);
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
          at::ScalarType::Half, at::ScalarType::BFloat16,
          self_2d.scalar_type(), "mha_fill", [&]() {
          mha_fill_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                  self_2d.data_ptr<scalar_t>(),
                  static_cast<int32_t*>(start_index.data_ptr()),
                  max_tokens);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
}

// extract seed and offset from PhiloxCudaState
__global__ void unpack(at::PhiloxCudaState arg, int64_t* rng_state_ptr) {
  if (arg.captured_) {
    rng_state_ptr[0] = static_cast<int64_t>(*arg.seed_.ptr);
    rng_state_ptr[1] = static_cast<int64_t>(
                    *(arg.offset_.ptr) + static_cast<int64_t>(arg.offset_intragraph_));
  } else {
    rng_state_ptr[0] = static_cast<int64_t>(arg.seed_.val);
    rng_state_ptr[1] = static_cast<int64_t>(arg.offset_.val);
  }
}

// extract PhiloxCudaState from CUDA random number generator
at::PhiloxCudaState init_philox_state(
                at::CUDAGeneratorImpl* gen,
                size_t elts_per_thread) {
  at::PhiloxCudaState philox_args;
  std::lock_guard<std::mutex> lock(gen->mutex_);
  philox_args = gen->philox_cuda_state(elts_per_thread);
  return philox_args;
}

// fused attention FWD with packed QKV
std::vector<at::Tensor> fused_attn_fwd_qkvpacked(
                size_t max_seqlen, bool is_training, float attn_scale,
                float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto qkv_sizes = QKV.sizes().vec();
  std::vector<size_t> qkv_shape{qkv_sizes.begin(), qkv_sizes.end()};
  std::vector<size_t> q_shape;
  for (auto i : qkv_shape) {
    if (i != 3) {
      q_shape.push_back(i);
    }
  }
  std::vector<int64_t> o_shape{q_shape.begin(), q_shape.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty(o_shape, options);

  // construct NVTE tensors
  TensorWrapper te_QKV, te_S, te_O, te_Bias, te_cu_seqlens;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_sizes = cu_seqlens.sizes().vec();
  std::vector<size_t> cu_seqlens_shape{cu_seqlens_sizes.begin(), cu_seqlens_sizes.end()};
  te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), cu_seqlens_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract random number generator seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed QKV
std::vector<at::Tensor> fused_attn_bwd_qkvpacked(
                size_t max_seqlen, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto qkv_sizes = QKV.sizes().vec();
  std::vector<size_t> qkv_shape{qkv_sizes.begin(), qkv_sizes.end()};
  std::vector<size_t> q_shape;
  for (auto i : qkv_shape) {
    if (i != 3) {
      q_shape.push_back(i);
    }
  }
  auto h = q_shape[q_shape.size() - 2];

  // create output tensor dQKV
  at::Tensor dQKV = torch::empty_like(QKV);
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
    dBias = torch::empty({1, static_cast<int64_t>(h),
                    static_cast<int64_t>(max_seqlen),
                    static_cast<int64_t>(max_seqlen)}, options);
    te_dBias = makeTransformerEngineTensor(dBias);
  }

  // construct NVTE tensors
  TensorWrapper te_QKV, te_O, te_dO, te_S, te_dP, te_dQKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(dQKV, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQKV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    nullptr, scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), qkv_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // convert auxiliary tensors from forward into NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_sizes = cu_seqlens.sizes().vec();
  std::vector<size_t> cu_seqlens_shape{cu_seqlens_sizes.begin(), cu_seqlens_sizes.end()};
  TensorWrapper te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), cu_seqlens_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQKV, dBias};
}

// fused attention FWD with packed KV
std::vector<at::Tensor> fused_attn_fwd_kvpacked(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto kv_sizes = KV.sizes().vec();
  std::vector<size_t> kv_shape{kv_sizes.begin(), kv_sizes.end()};
  std::vector<int64_t> o_shape{q_shape.begin(), q_shape.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty(o_shape, options);

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_S, te_O, te_Bias, te_cu_seqlens_q, te_cu_seqlens_kv;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed KV
std::vector<at::Tensor> fused_attn_bwd_kvpacked(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto kv_sizes = KV.sizes().vec();
  std::vector<size_t> kv_shape{kv_sizes.begin(), kv_sizes.end()};
  std::vector<size_t> k_shape;
  for (auto i : kv_shape) {
    if (i != 2) {
      k_shape.push_back(i);
    }
  }
  auto h_q = q_shape[q_shape.size() - 2];
  auto h_kv = k_shape[k_shape.size() - 2];
  auto d = q_shape[q_shape.size() - 1];

  // create output tensors dQ and dKV
  at::Tensor dQ = torch::empty_like(Q);
  at::Tensor dKV = torch::empty_like(KV);
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
    dBias = torch::empty({1, static_cast<int64_t>(h_q),
                    static_cast<int64_t>(max_seqlen_q),
                    static_cast<int64_t>(max_seqlen_kv)}, options);
    te_dBias = makeTransformerEngineTensor(dBias);
  }

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_O, te_dO, te_S, te_dP, te_dQ, te_dKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if (set_zero && ((h_q * d)% block_size == 0) && ((h_kv * d)% block_size == 0)) {
      mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dKV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dKV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                    scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), kv_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dKV, dBias};
}

// fused attention FWD with separate Q, K and V tensors
std::vector<at::Tensor> fused_attn_fwd(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor K,
                const at::Tensor V,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto k_sizes = K.sizes().vec();
  std::vector<size_t> k_shape{k_sizes.begin(), k_sizes.end()};
  auto v_sizes = V.sizes().vec();
  std::vector<size_t> v_shape{v_sizes.begin(), v_sizes.end()};

  // create output tensor O
  auto O = torch::empty_like(Q);

  // construct NVTE tensors
  TensorWrapper te_Q, te_K, te_V, te_S, te_O, te_Bias;
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto options = torch::TensorOptions().dtype(torch::kInt64).device(torch::kCUDA);
  auto rng_state = torch::empty({2}, options);
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with separate Q, K and V
std::vector<at::Tensor> fused_attn_bwd(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor K,
                const at::Tensor V,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto k_sizes = K.sizes().vec();
  std::vector<size_t> k_shape{k_sizes.begin(), k_sizes.end()};
  auto v_sizes = V.sizes().vec();
  std::vector<size_t> v_shape{v_sizes.begin(), v_sizes.end()};
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);

  at::Tensor dQ;
  at::Tensor dK;
  at::Tensor dV;
  at::Tensor dQKV, dKV;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  std::vector<int64_t> tmp_shape;
  switch (layout_group) {
      case NVTE_QKV_Layout_Group::NVTE_3HD:
          tmp_shape = std::vector<int64_t>{q_sizes.begin(), q_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(3));
          dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          break;
      case NVTE_QKV_Layout_Group::NVTE_H3D:
          tmp_shape = std::vector<int64_t>{q_sizes.begin(), q_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(3));
          dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_2HD:
          dQ = torch::empty_like(Q);
          tmp_shape = std::vector<int64_t>{k_sizes.begin(), k_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(2));
          dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_H2D:
          dQ = torch::empty_like(Q);
          tmp_shape = std::vector<int64_t>{k_sizes.begin(), k_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(2));
          dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_HD_HD:
          dQ = torch::empty_like(Q);
          dK = torch::empty_like(K);
          dV = torch::empty_like(V);
          break;
      default:
          NVTE_ERROR("QKV layout not supported!");
    }

  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
    dBias = torch::empty({1, static_cast<int64_t>(Q.size(-2)),
                    static_cast<int64_t>(max_seqlen_q),
                    static_cast<int64_t>(max_seqlen_kv)}, options);
    te_dBias = makeTransformerEngineTensor(dBias);
  }

  // construct NVTE tensors
  TensorWrapper te_Q, te_K, te_V, te_O, te_dO, te_S, te_dP, te_dQ, te_dK, te_dV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h_q = q_shape[q_shape.size() - 2];
    auto h_kv = k_shape[k_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero
          && ((h_q * d) % block_size == 0)
          && ((h_kv * d) % block_size == 0)
          && dQ.is_contiguous()
          && dK.is_contiguous()
          && dV.is_contiguous()) {
      mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dK, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dK.fill_(0);
      dV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                    scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dK = makeTransformerEngineTensor(dK.data_ptr(), k_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dV = makeTransformerEngineTensor(dV.data_ptr(), v_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dK = makeTransformerEngineTensor(dK.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dV = makeTransformerEngineTensor(dV.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dK.data(),
                  te_dV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dK.data(),
                  te_dV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dK, dV, dBias};
}

namespace flash_attention {

constexpr int warp_size = 32;
constexpr int type_size = 2;  // FP16 or BF16
constexpr int nvec = sizeof(uint64_t) / type_size;
constexpr int load_size = warp_size * nvec;
constexpr int block_size = 512;

template <typename T>
__launch_bounds__(block_size)
__global__ void prepare_kernel_fwd(const T *qkvi,
                                   T *qkv,
                                   const size_t B,
                                   const size_t S,
                                   const size_t Z,
                                   const size_t W) {
    const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
    const int id_in_warp = threadIdx.x % warp_size;
    const size_t offset_input = blockIdx.y * W + warpid * 3 * W * Z + id_in_warp * nvec;
    const T *my_input = qkvi + offset_input;

    const size_t s = warpid / B;
    if (s >= S) return;

    const size_t b = warpid % B;

    const size_t offset_output = blockIdx.y * B * S * Z * W +
                                 (s + b * S) * W * Z +
                                 id_in_warp * nvec;

    T *my_output = qkv + offset_output;

    for (int i = 0; i < Z; ++i) {
        uint64_t *out = reinterpret_cast<uint64_t*>(my_output + i * load_size);
        *out = *reinterpret_cast<const uint64_t*>(my_input + i * load_size * 3);
    }
}

template <typename T>
__launch_bounds__(block_size)
__global__ void prepare_kernel_bwd(const T *q, const T *k, const T *v,
                                   T *qkv, const size_t B, const size_t S,
                                   const size_t Z, const size_t W) {
    const T *input = blockIdx.y == 0 ? q : (blockIdx.y == 1 ? k : v);

    const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
    const int id_in_warp = threadIdx.x % warp_size;
    const size_t offset_input = warpid * W * Z + id_in_warp * nvec;
    const T *my_input = input + offset_input;

    const size_t b = warpid / S;
    if (b >= B) return;

    const size_t s = warpid % S;

    const size_t offset_output = (b + s * B) * 3 * W * Z +
                                 id_in_warp * nvec + blockIdx.y * W;

    T *my_output = qkv + offset_output;

    for (int i = 0; i < Z; ++i) {
        uint64_t *out = reinterpret_cast<uint64_t*>(my_output + i * load_size * 3);
        *out = *reinterpret_cast<const uint64_t*>(my_input + i * load_size);
    }
}

}  // namespace flash_attention

at::Tensor fa_prepare_fwd(at::Tensor qkvi) {
    NVTE_CHECK(qkvi.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(qkvi.scalar_type() == at::ScalarType::Half ||
               qkvi.scalar_type() == at::ScalarType::BFloat16);
    NVTE_CHECK(qkvi.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(qkvi.size(3) == flash_attention::load_size);
    NVTE_CHECK(qkvi.stride(3) == 1, "Wrong stride.");
    NVTE_CHECK(qkvi.stride(2) == 3 * qkvi.size(3), "Wrong stride.");
    NVTE_CHECK(qkvi.stride(1) == 3 * qkvi.size(3) * qkvi.size(2), "Wrong stride.");
    NVTE_CHECK(qkvi.stride(0) == 3 * qkvi.size(3) * qkvi.size(2) * qkvi.size(1), "Wrong stride.");

    // [s, b, n, h * 3] -> [3, b, s, n, h]
    std::vector<int64_t> shape = {3, qkvi.size(1), qkvi.size(0), qkvi.size(2), qkvi.size(3)};
    at::Tensor qkv = at::empty(shape, at::CUDA(qkvi.scalar_type()));

    size_t warps = qkvi.size(0) * qkvi.size(1);
    size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
    size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
    dim3 grid(blocks, 3);
    int threads = flash_attention::block_size;
    if (qkvi.scalar_type() == at::ScalarType::Half) {
        using dtype = at::Half;
        flash_attention::prepare_kernel_fwd<dtype><<<grid, threads, 0,
                                                     at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            shape[1],
            shape[2],
            shape[3],
            shape[4]);
    } else {
        using dtype = at::BFloat16;
        flash_attention::prepare_kernel_fwd<dtype><<<grid, threads, 0,
                                                     at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            shape[1],
            shape[2],
            shape[3],
            shape[4]);
    }

    return qkv;
}

at::Tensor fa_prepare_bwd(at::Tensor q, at::Tensor k, at::Tensor v) {
    NVTE_CHECK(q.is_contiguous());
    NVTE_CHECK(k.is_contiguous());
    NVTE_CHECK(v.is_contiguous());
    NVTE_CHECK(q.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(k.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(v.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(q.scalar_type() == at::ScalarType::Half ||
               q.scalar_type() == at::ScalarType::BFloat16);
    NVTE_CHECK(k.scalar_type() == q.scalar_type());
    NVTE_CHECK(v.scalar_type() == q.scalar_type());
    NVTE_CHECK(q.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(q.size(3) == flash_attention::load_size);
    NVTE_CHECK(k.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(k.size(3) == flash_attention::load_size);
    NVTE_CHECK(v.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(v.size(3) == flash_attention::load_size);

    // 3 x [s, b, n, h] -> [b, s, n, 3 * h]

    std::vector<int64_t> shape = {q.size(1), q.size(0), q.size(2), 3 * q.size(3)};
    at::Tensor qkv = at::empty(shape, at::CUDA(q.scalar_type()));

    size_t warps = q.size(0) * q.size(1);
    size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
    size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
    dim3 grid(blocks, 3);
    int threads = flash_attention::block_size;
    if (q.scalar_type() == at::ScalarType::Half) {
        using dtype = at::Half;
        flash_attention::prepare_kernel_bwd<dtype><<<grid, threads, 0,
                                                 at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(),
            k.data_ptr<dtype>(),
            v.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            q.size(0),
            q.size(1),
            q.size(2),
            q.size(3));
    } else {
        using dtype = at::BFloat16;
        flash_attention::prepare_kernel_bwd<dtype><<<grid, threads, 0,
                                                 at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(),
            k.data_ptr<dtype>(),
            v.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            q.size(0),
            q.size(1),
            q.size(2),
            q.size(3));
    }

    return qkv;
}

// LSE correction kernel for Context Parallel
__forceinline__
__device__ int binary_search(int target, int *array, int len) {
  int left = 1, right = len - 1;
  while (left < right) {
    int mid = (left + right) / 2;
    if (array[mid] <= target) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }
  return left - 1;
}

__global__ void LseCorrectionKernel(double *lse, float *lse_per_step, int *cu_seqlens,
                                    int batch, int num_heads, int max_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  //batch + 1 = len(cu_seqlens)
  if (threadIdx.x < batch + 1) {
    cu_seqlens_s[threadIdx.x] = cu_seqlens[threadIdx.x] / 2;
  }
  __syncthreads();

  int token_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (token_id >= cu_seqlens_s[batch]) {
    return;
  }

  int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
  for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
    // Calculate indices
    int row = seq_id * num_heads + head_id;
    int col = token_id - cu_seqlens_s[seq_id];
    int len_per_step = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];

    // Load from global memory
    double val = lse[row * max_seqlen + col + len_per_step];
    float val_per_step = lse_per_step[row * max_seqlen / 2 + col];

    // Correction
    val = log(exp(val) + exp((double)val_per_step));

    // Write to global memory
    lse[row * max_seqlen + col + len_per_step] = val;
  }
}

void lse_correction(at::Tensor &lse, const at::Tensor &lse_per_step, const at::Tensor &cu_seqlens_per_step,
                    int batch, int num_heads, int max_seqlen, int total_tokens, int num_sms) {
  auto lse_type = lse.scalar_type();
  auto lse_per_step_type = lse_per_step.scalar_type();
  auto cu_seqlens_per_step_type = cu_seqlens_per_step.scalar_type();
  NVTE_CHECK(lse_type == at::ScalarType::Double, "lse should be double");
  NVTE_CHECK(lse_per_step_type == at::ScalarType::Float, "lse_per_step should be float");
  NVTE_CHECK(cu_seqlens_per_step_type == at::ScalarType::Int, "cu_seqlens should be int");
  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = (num_sms * 2 + grid_x - 1) / grid_x;
  dim3 grid = {grid_x, grid_y, 1};
  LseCorrectionKernel<<<grid, block, (batch + 1) * sizeof(int), at::cuda::getCurrentCUDAStream()>>>(
    (double*)lse.data_ptr(), (float*)lse_per_step.data_ptr(), (int*)cu_seqlens_per_step.data_ptr(), batch, num_heads, max_seqlen);
}

template <int second_half>
__global__ void cp_thd_read_half_tensor_kernel(void *half_tensor, void *tensor, int *cu_seqlens,
                                               int cu_seqlens_size, int hidden_size_in_bytes) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i < cu_seqlens_size; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int warpid = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  int laneid = threadIdx.x % 32;
  int num_warps = (blockDim.x * gridDim.x) / 32;
  int num_total_tokens = cu_seqlens_s[cu_seqlens_size - 1];
  int num_float4s_per_token = hidden_size_in_bytes / sizeof(float4);

  size_t offset = num_total_tokens * (size_t)hidden_size_in_bytes;
  half_tensor = (void*)((char*)half_tensor + offset * blockIdx.y);
  tensor = (void*)((char*)tensor + 2 * offset * blockIdx.y);

  for (int token_id = warpid; token_id < num_total_tokens; token_id += num_warps) {
    int seqid = binary_search(token_id, cu_seqlens_s, cu_seqlens_size);
    float4* cur_half_token = (float4*)((char*)half_tensor + token_id * (size_t)hidden_size_in_bytes);
    float4* cur_token = (float4*)((char*)tensor + (token_id + cu_seqlens_s[seqid + second_half]) * (size_t)hidden_size_in_bytes);
    for (int idx = laneid; idx < num_float4s_per_token; idx += 32) {
      cur_half_token[idx] = cur_token[idx];
    }
  }
}

at::Tensor cp_thd_read_half_tensor(const at::Tensor &input, const at::Tensor &cu_seqlens, int seq_dim, bool second_half) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int, "cu_seqlens should be int");
  size_t hidden_size_in_bytes = c10::elementSize(input.scalar_type()) * input.size(seq_dim + 1) * input.size(seq_dim + 2);
  NVTE_CHECK(hidden_size_in_bytes % 16 == 0, "hidden_size_in_bytes % 16 should be 0");

  // Generate output
  std::vector<int64_t> shape(input.dim());
  for (size_t i = 0; i < shape.size(); i++) {
    shape[i] = input.size(i);
  }
  shape[seq_dim] /= 2;
  at::Tensor output = at::empty(shape, at::CUDA(input.scalar_type()));

  // Launch Kernel
  constexpr unsigned int block = 256;
  unsigned int grid_x = (input.size(seq_dim) / 2 * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  if (seq_dim != 0) {
    grid_y = input.size(0);
  }
  dim3 grid = {grid_x, grid_y};
  if (second_half) {
    cp_thd_read_half_tensor_kernel<1><<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
      (void*)output.data_ptr(), (void*)input.data_ptr(), (int*)cu_seqlens.data_ptr(), cu_seqlens.size(0), hidden_size_in_bytes);
  } else {
    cp_thd_read_half_tensor_kernel<0><<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
      (void*)output.data_ptr(), (void*)input.data_ptr(), (int*)cu_seqlens.data_ptr(), cu_seqlens.size(0), hidden_size_in_bytes);
  }

  return output;
}

template <typename dtype, int group_size>
__global__ void out_correction_thd_kernel(dtype *out, dtype *out_per_step,
                                          float *lse, float *lse_per_step,
                                          int *cu_seqlens, int batch,
                                          int num_heads, int dim_per_head,
                                          int max_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i];
  }
  __syncthreads();

  int warpid = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int laneid = threadIdx.x % group_size;
  int num_warps = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[batch];

  int num_inner_loops = dim_per_head * sizeof(dtype) / sizeof(float4);

  for (int token_id = warpid; token_id < num_total_tokens; token_id += num_warps) {
    int seqid = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      int row = seqid * num_heads + head_id;
      int col = token_id - cu_seqlens_s[seqid];
      size_t idx = (size_t)row * max_seqlen + col;
      float lse_corrected_exp = exp(lse_per_step[idx] - lse[idx]);
      idx = ((size_t)token_id * num_heads + head_id) * dim_per_head;
      dtype *cur_out = out + idx;
      dtype *cur_out_per_step = out_per_step + idx;

      for (int j = laneid; j < num_inner_loops; j += group_size) {
        float4 tmp_out_per_step = ((float4*)cur_out_per_step)[j];
        float4 tmp_out = ((float4*)cur_out)[j];
        dtype *tmp_out_per_step_p = (dtype*)&tmp_out_per_step;
        dtype *tmp_out_p = (dtype*)&tmp_out;
        for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
          tmp_out_p[i] += tmp_out_per_step_p[i] * lse_corrected_exp;
        }
        ((float4*)cur_out)[j] = tmp_out;
        // cur_out[j] += cur_out_per_step[j] * lse_corrected_exp;
      }
    }
  }
}

void out_correction_thd(at::Tensor &out, const at::Tensor &out_per_step,
                        const at::Tensor &lse, const at::Tensor &lse_per_step,
                        const at::Tensor &cu_seqlens) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int, "cu_seqlens should be int");
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float, "lse should be float");
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float, "lse_per_step should be float");
  NVTE_CHECK(out.scalar_type() == out_per_step.scalar_type(), "type of out and out_per_step should be the same");

  int batch = lse.size(0);
  unsigned int num_heads = lse.size(1);
  int max_seqlen = lse.size(2);
  int total_tokens = out.size(0);
  int dim_per_head = out.size(2);

  NVTE_CHECK(out.size(1) == num_heads, "out.size(1) != num_heads");
  // NVTE_CHECK(out_per_step.size(0) == num_heads, "out.size(1) != num_heads");
  NVTE_CHECK(out_per_step.size(1) == num_heads, "out.size(1) != num_heads");
  NVTE_CHECK(out_per_step.size(2) == dim_per_head, "out_per_step.size(2) != dim_per_head");
  NVTE_CHECK(lse_per_step.size(0) == batch, "error 4");
  NVTE_CHECK(lse_per_step.size(1) == num_heads, "error 5");
  NVTE_CHECK(lse_per_step.size(2) == max_seqlen, "error 6");
  NVTE_CHECK(cu_seqlens.size(0) == batch + 1, "error 7");

  constexpr int group_size = 16;
  constexpr unsigned int block = 512;
  unsigned int grid_x = min((total_tokens * group_size + block - 1) / block, 256);
  dim3 grid = {grid_x, num_heads};

  if (out.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    out_correction_thd_kernel<dtype, group_size><<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
      out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(), lse_per_step.data_ptr<float>(),
      cu_seqlens.data_ptr<int>(), batch, num_heads, dim_per_head, max_seqlen);
  } else if (out.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    out_correction_thd_kernel<dtype, group_size><<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
      out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(), lse_per_step.data_ptr<float>(),
      cu_seqlens.data_ptr<int>(), batch, num_heads, dim_per_head, max_seqlen);
  } else if (out.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    out_correction_thd_kernel<dtype, group_size><<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
      out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(), lse_per_step.data_ptr<float>(),
      cu_seqlens.data_ptr<int>(), batch, num_heads, dim_per_head, max_seqlen);
  } else {
    NVTE_ERROR("unsupported dtype of out\n");
  }
}

template <typename dtype, int group_size>
__global__ void out_correction_thd_kernel2(dtype *out, dtype *out_per_step,
                                           float *lse, float *lse_per_step,
                                           int *cu_seqlens, int batch,
                                           int num_heads, int dim_per_head,
                                           int max_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int warpid = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int laneid = threadIdx.x % group_size;
  int num_warps = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[batch];

  int num_inner_loops = dim_per_head * sizeof(dtype) / sizeof(float4);

  for (int token_id = warpid; token_id < num_total_tokens; token_id += num_warps) {
    int seqid = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      int row = seqid * num_heads + head_id;
      int col = token_id - cu_seqlens_s[seqid];
      int len = cu_seqlens_s[seqid + 1] - cu_seqlens_s[seqid];

      float val = lse[row * max_seqlen + col + len];
      float val_per_step = lse_per_step[row * max_seqlen / 2 + col];
      float lse_corrected_exp = exp(val_per_step - val);

      dtype *cur_out = out + ((token_id + cu_seqlens_s[seqid+1]) * num_heads + head_id) * dim_per_head;
      dtype *cur_out_per_step = out_per_step + (token_id * num_heads + head_id) * dim_per_head;

      for (int j = laneid; j < num_inner_loops; j += group_size) {
        float4 tmp_out_per_step = ((float4*)cur_out_per_step)[j];
        float4 tmp_out = ((float4*)cur_out)[j];
        dtype *tmp_out_per_step_p = (dtype*)&tmp_out_per_step;
        dtype *tmp_out_p = (dtype*)&tmp_out;
        for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
          tmp_out_p[i] += tmp_out_per_step_p[i] * lse_corrected_exp;
        }
        ((float4*)cur_out)[j] = tmp_out;
      }
    }
  }
}

void out_correction_thd2(at::Tensor &out, const at::Tensor &out_per_step,
                         const at::Tensor &lse, const at::Tensor &lse_per_step,
                         const at::Tensor &cu_seqlens) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int, "cu_seqlens should be int");
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float, "lse should be float");
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float, "lse_per_step should be float");
  NVTE_CHECK(out.scalar_type() == out_per_step.scalar_type(), "type of out and out_per_step should be the same");

  int batch = lse.size(0);
  unsigned int num_heads = lse.size(1);
  int max_seqlen = lse.size(2);
  int total_tokens = out.size(0);
  int dim_per_head = out.size(2);

  NVTE_CHECK(out.size(1) == num_heads, "out.size(1) != num_heads");
  NVTE_CHECK(out_per_step.size(1) == num_heads, "out.size(1) != num_heads");
  NVTE_CHECK(out_per_step.size(2) == dim_per_head, "out_per_step.size(2) != dim_per_head");
  NVTE_CHECK(lse_per_step.size(0) == batch, "error 4");
  NVTE_CHECK(lse_per_step.size(1) == num_heads, "error 5");
  NVTE_CHECK(lse_per_step.size(2) == max_seqlen/2, "error 6");
  NVTE_CHECK(cu_seqlens.size(0) == batch + 1, "error 7");

  constexpr unsigned int block = 512;
  constexpr int group_size = 16;
  unsigned int grid_x = min((total_tokens / 2 * group_size + block - 1) / block, 256);
  dim3 grid = {grid_x, num_heads};

  if (out.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    out_correction_thd_kernel2<dtype, group_size><<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
      out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(), lse_per_step.data_ptr<float>(),
      cu_seqlens.data_ptr<int>(), batch, num_heads, dim_per_head, max_seqlen);
  } else if (out.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    out_correction_thd_kernel2<dtype, group_size><<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
      out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(), lse_per_step.data_ptr<float>(),
      cu_seqlens.data_ptr<int>(), batch, num_heads, dim_per_head, max_seqlen);
  } else if (out.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    out_correction_thd_kernel2<dtype, group_size><<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
      out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(), lse_per_step.data_ptr<float>(),
      cu_seqlens.data_ptr<int>(), batch, num_heads, dim_per_head, max_seqlen);
  } else {
    NVTE_ERROR("unsupported dtype of out\n");
  }
}

__global__ void cp_thd_bwd_lse_kernel(float *half_lse, float *lse, int *cu_seqlens,
                                      int batch, int num_heads, int max_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  //batch + 1 = len(cu_seqlens)
  if (threadIdx.x < batch + 1) {
    cu_seqlens_s[threadIdx.x] = cu_seqlens[threadIdx.x] / 2;
  }
  __syncthreads();

  int token_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (token_id >= cu_seqlens_s[batch]) {
    return;
  }

  int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
  for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
    // Calculate indices
    int row = seq_id * num_heads + head_id;
    int col = token_id - cu_seqlens_s[seq_id];
    int len_per_step = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];

    // Load from global memory
    double val = lse[row * max_seqlen + col + len_per_step];

    half_lse[row*max_seqlen/2 + col] = val;
  }
}

at::Tensor cp_thd_bwd_lse(const at::Tensor &lse, const at::Tensor &cu_seqlens, int total_tokens) {
  auto lse_type = lse.scalar_type();
  auto cu_seqlens_type = cu_seqlens.scalar_type();
  NVTE_CHECK(lse_type == at::ScalarType::Float, "lse should be float");
  NVTE_CHECK(cu_seqlens_type == at::ScalarType::Int, "cu_seqlens should be int");

  // Generate output
  std::vector<int64_t> shape(lse.dim());
  for (size_t i = 0; i < shape.size(); i++) {
    shape[i] = lse.size(i);
  }
  shape[2] /= 2;
  at::Tensor half_lse = at::zeros(shape, at::CUDA(lse.scalar_type()));

  int batch = lse.size(0);
  int num_heads = lse.size(1);
  int max_seqlen = lse.size(2);

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = (108 * 2 + grid_x - 1) / grid_x;
  dim3 grid = {grid_x, grid_y, 1};
  cp_thd_bwd_lse_kernel<<<grid, block, (batch + 1) * sizeof(int), at::cuda::getCurrentCUDAStream()>>>(
    half_lse.data_ptr<float>(), lse.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch, num_heads, max_seqlen);

  return half_lse;
}

__global__ void generate_thd_indices_for_cp_kernel(int *output, int *cu_seqlens, int batch) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  int num_total_tokens = cu_seqlens_s[batch];

  output += num_total_tokens * blockIdx.y;

  for (int i = tid; i < num_total_tokens; i += num_threads) {
    int seqid = binary_search(i, cu_seqlens_s, batch + 1);
    output[i] = cu_seqlens_s[seqid + blockIdx.y] + i;
  }
}

at::Tensor generate_thd_indices_for_cp(const at::Tensor &cu_seqlens, int total_tokens) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int, "cu_seqlens should be int");
  NVTE_CHECK(total_tokens % 2 == 0, "total_tokens % 2 should be 0");

  std::vector<int64_t> shape(2);
  shape[0] = 2;
  shape[1] = total_tokens / 2;
  at::Tensor output = at::empty(shape, at::CUDA(cu_seqlens.scalar_type()));

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = 2;
  dim3 grid = {grid_x, grid_y};
  generate_thd_indices_for_cp_kernel<<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
    output.data_ptr<int>(), cu_seqlens.data_ptr<int>(), cu_seqlens.size(0) - 1);

  return output;
}

template <typename dtype, int second_half, int add, int group_size>
__global__ void thd_rw_half_kernel(dtype *whole, dtype *half_tensor, int *cu_seqlens,
                                   int cu_seqlens_size, int hidden_size) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i < cu_seqlens_size; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int group_id = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int lane_id = threadIdx.x % group_size;
  int num_groups = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[cu_seqlens_size - 1];

  size_t offset = num_total_tokens * (size_t)hidden_size;
  half_tensor = half_tensor + offset * blockIdx.y;
  whole = whole + 2 * offset * blockIdx.y;

  int num_inner_loops = hidden_size * sizeof(dtype) / sizeof(float4);

  for (int token_id = group_id; token_id < num_total_tokens; token_id += num_groups) {
    int seq_id = binary_search(token_id, cu_seqlens_s, cu_seqlens_size);
    dtype *cur_half_token = half_tensor + token_id * (size_t)hidden_size;
    dtype *cur_token = whole + (token_id + cu_seqlens_s[seq_id + second_half]) * (size_t)hidden_size;
    for (int idx = lane_id; idx < num_inner_loops; idx += group_size) {
      float4 tmp_half_token = ((float4*)cur_half_token)[idx];
      float4 tmp_token;
      if constexpr (add == 1) {
        tmp_token = ((float4*)cur_token)[idx];
        dtype *tmp_half_token_p = (dtype*)(&tmp_half_token);
        dtype *tmp_token_p = (dtype*)(&tmp_token);
        for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
          tmp_token_p[i] += tmp_half_token_p[i];
        }
      } else {
        tmp_token = tmp_half_token;
      }
      ((float4*)cur_token)[idx] = tmp_token;
    }
  }
}

template <typename dtype>
void helper(at::Tensor &whole, const at::Tensor &half, const at::Tensor &cu_seqlens, int second_half, int add, int seq_dim) {
  size_t hidden_size = whole.size(seq_dim + 1) * whole.size(seq_dim + 2);
  NVTE_CHECK((hidden_size * c10::elementSize(whole.scalar_type())) % 16 == 0, "helper error");

  // Launch Kernel
  constexpr unsigned int block = 256;
  unsigned int grid_x = (whole.size(seq_dim) / 2 * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  if (seq_dim != 0) {
    grid_y = whole.size(0);
  }
  dim3 grid = {grid_x, grid_y};

  if (second_half == 0 && add == 0) {
    thd_rw_half_kernel<dtype, 0, 0, 32><<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
      whole.data_ptr<dtype>(), half.data_ptr<dtype>(), cu_seqlens.data_ptr<int>(), cu_seqlens.size(0), hidden_size);
  } else if (second_half == 0 && add == 1) {
    thd_rw_half_kernel<dtype, 0, 1, 32><<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
      whole.data_ptr<dtype>(), half.data_ptr<dtype>(), cu_seqlens.data_ptr<int>(), cu_seqlens.size(0), hidden_size);
  } else if (second_half == 1 && add == 0) {
    thd_rw_half_kernel<dtype, 1, 0, 32><<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
      whole.data_ptr<dtype>(), half.data_ptr<dtype>(), cu_seqlens.data_ptr<int>(), cu_seqlens.size(0), hidden_size);
  } else if (second_half == 1 && add == 1) {
    thd_rw_half_kernel<dtype, 1, 1, 32><<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
      whole.data_ptr<dtype>(), half.data_ptr<dtype>(), cu_seqlens.data_ptr<int>(), cu_seqlens.size(0), hidden_size);
  } else {
    NVTE_ERROR("thd_op1_helper Error\n");
  }
}

void thd_op1(at::Tensor &whole, const at::Tensor &half, const at::Tensor &cu_seqlens, int second_half, int add, int seq_dim) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int, "cu_seqlens should be int");

  if (whole.scalar_type() == at::ScalarType::Half) {
    helper<at::Half>(whole, half, cu_seqlens, second_half, add, seq_dim);
  } else if (whole.scalar_type() == at::ScalarType::BFloat16) {
    helper<at::BFloat16>(whole, half, cu_seqlens, second_half, add, seq_dim);
  } else if (whole.scalar_type() == at::ScalarType::Float) {
    helper<float>(whole, half, cu_seqlens, second_half, add, seq_dim);
  } else {
    NVTE_ERROR("thd_op1_error\n");
  }
}

template <typename dtype, int left_add, int group_size>
__global__ void thd_rw_half_kernel2(dtype *whole, dtype *t2, int *cu_seqlens,
                                    int cu_seqlens_size, int hidden_size) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i < cu_seqlens_size; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i];
  }
  __syncthreads();

  int group_id = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int lane_id = threadIdx.x % group_size;
  int num_groups = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[cu_seqlens_size - 1];

  size_t offset = num_total_tokens * (size_t)hidden_size;
  t2 = t2 + offset * blockIdx.y;
  whole = whole + offset * blockIdx.y;

  int num_inner_loops = hidden_size * sizeof(dtype) / sizeof(float4);

  for (int token_id = group_id; token_id < num_total_tokens; token_id += num_groups) {
    int seq_id = binary_search(token_id, cu_seqlens_s, cu_seqlens_size);
    int len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
    bool is_left = (token_id - cu_seqlens[seq_id]) < (len / 2);

    dtype *cur_half_token = t2 + token_id * (size_t)hidden_size;
    dtype *cur_token = whole + token_id * (size_t)hidden_size;
    for (int idx = lane_id; idx < num_inner_loops; idx += group_size) {
      float4 tmp_half_token = ((float4*)cur_half_token)[idx];
      float4 tmp_token;
      if ((is_left && left_add == 1) || (!is_left && left_add == 0)) {
        tmp_token = ((float4*)cur_token)[idx];
        dtype *tmp_half_token_p = (dtype*)(&tmp_half_token);
        dtype *tmp_token_p = (dtype*)(&tmp_token);
        for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
          tmp_token_p[i] += tmp_half_token_p[i];
        }
      } else {
        tmp_token = tmp_half_token;
      }
      ((float4*)cur_token)[idx] = tmp_token;
    }
  }
}

template <typename dtype>
void helper2(at::Tensor &whole, const at::Tensor &half, const at::Tensor &cu_seqlens, int add, int seq_dim) {
  size_t hidden_size = whole.size(seq_dim + 1) * whole.size(seq_dim + 2);
  NVTE_CHECK((hidden_size * c10::elementSize(whole.scalar_type())) % 16 == 0, "helper error");

  // Launch Kernel
  constexpr unsigned int block = 256;
  unsigned int grid_x = (whole.size(seq_dim) * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  if (seq_dim != 0) {
    grid_y = whole.size(0);
  }
  dim3 grid = {grid_x, grid_y};

  if (add == 0) {
    thd_rw_half_kernel2<dtype, 0, 32><<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
      whole.data_ptr<dtype>(), half.data_ptr<dtype>(), cu_seqlens.data_ptr<int>(), cu_seqlens.size(0), hidden_size);
  } else if (add == 1) {
    thd_rw_half_kernel2<dtype, 1, 32><<<grid, block, sizeof(int) * cu_seqlens.size(0), at::cuda::getCurrentCUDAStream()>>>(
      whole.data_ptr<dtype>(), half.data_ptr<dtype>(), cu_seqlens.data_ptr<int>(), cu_seqlens.size(0), hidden_size);
  } else {
    NVTE_ERROR("thd_op1_helper Error\n");
  }
}

void thd_op2(at::Tensor &whole, const at::Tensor &half, const at::Tensor &cu_seqlens, int add, int seq_dim) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int, "cu_seqlens should be int");
  if (whole.scalar_type() == at::ScalarType::Half) {
    helper2<at::Half>(whole, half, cu_seqlens, add, seq_dim);
  } else if (whole.scalar_type() == at::ScalarType::BFloat16) {
    helper2<at::BFloat16>(whole, half, cu_seqlens, add, seq_dim);
  } else if (whole.scalar_type() == at::ScalarType::Float) {
    helper2<float>(whole, half, cu_seqlens, add, seq_dim);
  } else {
    NVTE_ERROR("thd_op1_error\n");
  }
}

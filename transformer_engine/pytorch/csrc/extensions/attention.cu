#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"

constexpr int block_size = 512;
constexpr int ctas_per_sm = 4;

// get the fused attention backend
NVTE_Fused_Attn_Backend get_fused_attn_backend(
                const transformer_engine::DType q_dtype,
                const transformer_engine::DType kv_dtype,
                NVTE_QKV_Layout qkv_layout,
                NVTE_Bias_Type bias_type,
                NVTE_Mask_Type attn_mask_type,
                float p_dropout,
                size_t num_attn_heads, size_t num_gqa_groups,
                size_t max_seqlen_q, size_t max_seqlen_kv,
                size_t head_dim) {
  NVTE_Fused_Attn_Backend fused_attention_backend =
          nvte_get_fused_attn_backend(
                          static_cast<NVTEDType>(q_dtype), static_cast<NVTEDType>(kv_dtype),
                          qkv_layout, bias_type, attn_mask_type, p_dropout,
                          num_attn_heads, num_gqa_groups,
                          max_seqlen_q, max_seqlen_kv, head_dim);
  return fused_attention_backend;
}

// fast zero-fills of tensors
template <typename scalar_t>
__global__ void __launch_bounds__(block_size) mha_fill_kernel(scalar_t* out_tensor,
                const int32_t* const start_row,
                const size_t num_rows) {
  size_t row_stride = gridDim.y * blockDim.x;
  size_t row_index = blockIdx.x + static_cast<size_t>(start_row[0]);
  size_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  while (row_index < num_rows) {
    out_tensor[row_index*row_stride + col_index] = 0;
    row_index += gridDim.x;
  }
}

// fast zero-fills of tensors
void mha_fill(const at::Tensor &self, const at::Tensor &start_index) {
  auto max_tokens = self.size(0);
  auto self_2d = self.view({max_tokens, -1});
  auto fcd_size = self_2d.size(1);
  TORCH_CHECK(self.is_contiguous(), "input not contiguous");
  TORCH_CHECK(fcd_size % block_size == 0, "input size not aligned to block size");
  const int num_mp = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  uint64_t num_blk_y = (uint64_t)(fcd_size / block_size);
  uint64_t num_blk_x = (uint64_t)((num_mp * ctas_per_sm + num_blk_y - 1) / num_blk_y);
  dim3 dim_grid(num_blk_x, num_blk_y);
  dim3 dim_block(block_size);
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
          at::ScalarType::Half, at::ScalarType::BFloat16,
          self_2d.scalar_type(), "mha_fill", [&]() {
          mha_fill_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                  self_2d.data_ptr<scalar_t>(),
                  static_cast<int32_t*>(start_index.data_ptr()),
                  max_tokens);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
}

// extract seed and offset from PhiloxCudaState
__global__ void unpack(at::PhiloxCudaState arg, int64_t* rng_state_ptr) {
  if (arg.captured_) {
    rng_state_ptr[0] = static_cast<int64_t>(*arg.seed_.ptr);
    rng_state_ptr[1] = static_cast<int64_t>(
                    *(arg.offset_.ptr) + static_cast<int64_t>(arg.offset_intragraph_));
  } else {
    rng_state_ptr[0] = static_cast<int64_t>(arg.seed_.val);
    rng_state_ptr[1] = static_cast<int64_t>(arg.offset_.val);
  }
}

// extract PhiloxCudaState from CUDA random number generator
at::PhiloxCudaState init_philox_state(
                at::CUDAGeneratorImpl* gen,
                size_t elts_per_thread) {
  at::PhiloxCudaState philox_args;
  std::lock_guard<std::mutex> lock(gen->mutex_);
  philox_args = gen->philox_cuda_state(elts_per_thread);
  return philox_args;
}

// fused attention FWD with packed QKV
std::vector<at::Tensor> fused_attn_fwd_qkvpacked(
                size_t max_seqlen, bool is_training, float attn_scale,
                float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto qkv_sizes = QKV.sizes().vec();
  std::vector<size_t> qkv_shape{qkv_sizes.begin(), qkv_sizes.end()};
  std::vector<size_t> q_shape;
  for (auto i : qkv_shape) {
    if (i != 3) {
      q_shape.push_back(i);
    }
  }
  std::vector<int64_t> o_shape{q_shape.begin(), q_shape.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty(o_shape, options);

  // construct NVTE tensors
  TensorWrapper te_QKV, te_S, te_O, te_Bias, te_cu_seqlens;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_sizes = cu_seqlens.sizes().vec();
  std::vector<size_t> cu_seqlens_shape{cu_seqlens_sizes.begin(), cu_seqlens_sizes.end()};
  te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), cu_seqlens_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract random number generator seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed QKV
std::vector<at::Tensor> fused_attn_bwd_qkvpacked(
                size_t max_seqlen, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto qkv_sizes = QKV.sizes().vec();
  std::vector<size_t> qkv_shape{qkv_sizes.begin(), qkv_sizes.end()};
  std::vector<size_t> q_shape;
  for (auto i : qkv_shape) {
    if (i != 3) {
      q_shape.push_back(i);
    }
  }
  auto h = q_shape[q_shape.size() - 2];

  // create output tensor dQKV
  at::Tensor dQKV = torch::empty_like(QKV);
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);

  // construct NVTE tensors
  TensorWrapper te_QKV, te_O, te_dO, te_S, te_dP, te_dQKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(dQKV, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQKV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    nullptr, scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), qkv_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), qkv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // convert auxiliary tensors from forward into NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS)
    && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h),
                    static_cast<int64_t>(max_seqlen),
                    static_cast<int64_t>(max_seqlen)}, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_sizes = cu_seqlens.sizes().vec();
  std::vector<size_t> cu_seqlens_shape{cu_seqlens_sizes.begin(), cu_seqlens_sizes.end()};
  TensorWrapper te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), cu_seqlens_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQKV, dBias};
}

// fused attention FWD with packed KV
std::vector<at::Tensor> fused_attn_fwd_kvpacked(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto kv_sizes = KV.sizes().vec();
  std::vector<size_t> kv_shape{kv_sizes.begin(), kv_sizes.end()};
  std::vector<int64_t> o_shape{q_shape.begin(), q_shape.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty(o_shape, options);

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_S, te_O, te_Bias, te_cu_seqlens_q, te_cu_seqlens_kv;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed KV
std::vector<at::Tensor> fused_attn_bwd_kvpacked(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto kv_sizes = KV.sizes().vec();
  std::vector<size_t> kv_shape{kv_sizes.begin(), kv_sizes.end()};
  std::vector<size_t> k_shape;
  for (auto i : kv_shape) {
    if (i != 2) {
      k_shape.push_back(i);
    }
  }
  auto h_q = q_shape[q_shape.size() - 2];
  auto h_kv = k_shape[k_shape.size() - 2];
  auto d = q_shape[q_shape.size() - 1];

  // create output tensors dQ and dKV
  at::Tensor dQ = torch::empty_like(Q);
  at::Tensor dKV = torch::empty_like(KV);
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_O, te_dO, te_S, te_dP, te_dQ, te_dKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if (set_zero && ((h_q * d)% block_size == 0) && ((h_kv * d)% block_size == 0)) {
      mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dKV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dKV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                    scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), kv_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), kv_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS)
    && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h_q),
                    static_cast<int64_t>(max_seqlen_q),
                    static_cast<int64_t>(max_seqlen_kv)}, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dKV, dBias};
}

// fused attention FWD with separate Q, K and V tensors
std::vector<at::Tensor> fused_attn_fwd(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor K,
                const at::Tensor V,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto k_sizes = K.sizes().vec();
  std::vector<size_t> k_shape{k_sizes.begin(), k_sizes.end()};
  auto v_sizes = V.sizes().vec();
  std::vector<size_t> v_shape{v_sizes.begin(), v_sizes.end()};

  // create output tensor O
  auto O = torch::empty_like(Q);

  // construct NVTE tensors
  TensorWrapper te_Q, te_K, te_V, te_S, te_O, te_Bias;
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0)) {
      mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto options = torch::TensorOptions().dtype(torch::kInt64).device(torch::kCUDA);
  auto rng_state = torch::empty({2}, options);
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
            if (i < nvte_aux_tensor_pack.size - 2) {
                output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
            } else if (i == nvte_aux_tensor_pack.size - 2) {
                output_tensor = rng_state;
            } else if (i == nvte_aux_tensor_pack.size - 1) {
                output_tensor = Bias.value();
            }
        } else {
            output_tensor = (i < nvte_aux_tensor_pack.size-1)
                ? allocateSpace(tensor->data.shape, tensor->data.dtype, false) : rng_state;
        }
    } else {
        output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with separate Q, K and V
std::vector<at::Tensor> fused_attn_bwd(
                size_t max_seqlen_q, size_t max_seqlen_kv,
                float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor K,
                const at::Tensor V,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto k_sizes = K.sizes().vec();
  std::vector<size_t> k_shape{k_sizes.begin(), k_sizes.end()};
  auto v_sizes = V.sizes().vec();
  std::vector<size_t> v_shape{v_sizes.begin(), v_sizes.end()};
  auto h_q = q_shape[q_shape.size() - 2];
  auto h_kv = k_shape[k_shape.size() - 2];
  auto d = q_shape[q_shape.size() - 1];
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);

  at::Tensor dQ;
  at::Tensor dK;
  at::Tensor dV;
  at::Tensor dQKV, dKV;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  std::vector<int64_t> tmp_shape;
  switch (layout_group) {
      case NVTE_QKV_Layout_Group::NVTE_3HD:
          tmp_shape = std::vector<int64_t>{q_sizes.begin(), q_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(3));
          dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          break;
      case NVTE_QKV_Layout_Group::NVTE_H3D:
          tmp_shape = std::vector<int64_t>{q_sizes.begin(), q_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(3));
          dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_2HD:
          dQ = torch::empty_like(Q);
          tmp_shape = std::vector<int64_t>{k_sizes.begin(), k_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(2));
          dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 3);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_H2D:
          dQ = torch::empty_like(Q);
          tmp_shape = std::vector<int64_t>{k_sizes.begin(), k_sizes.end()};
          tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(2));
          dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
          dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
              torch::indexing::Slice(0, torch::indexing::None, 1)}).squeeze(tmp_shape.size() - 2);
          break;
      case NVTE_QKV_Layout_Group::NVTE_HD_HD_HD:
          dQ = torch::empty_like(Q);
          dK = torch::empty_like(K);
          dV = torch::empty_like(V);
          break;
      default:
          NVTE_ERROR("QKV layout not supported!");
    }

  // construct NVTE tensors
  TensorWrapper te_Q, te_K, te_V, te_O, te_dO, te_S, te_dP, te_dQ, te_dK, te_dV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if (set_zero
          && ((h_q * d) % block_size == 0)
          && ((h_kv * d) % block_size == 0)
          && dQ.is_contiguous()
          && dK.is_contiguous()
          && dV.is_contiguous()) {
      mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dK, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dK.fill_(0);
      dV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                    scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dK = makeTransformerEngineTensor(dK.data_ptr(), k_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dV = makeTransformerEngineTensor(dV.data_ptr(), v_shape, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), q_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dK = makeTransformerEngineTensor(dK.data_ptr(), k_shape,
                    qkv_type, nullptr, nullptr, nullptr);
    te_dV = makeTransformerEngineTensor(dV.data_ptr(), v_shape,
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS)
    && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h_q),
                    static_cast<int64_t>(max_seqlen_q),
                    static_cast<int64_t>(max_seqlen_kv)}, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dK.data(),
                  te_dV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd(
                  te_Q.data(),
                  te_K.data(),
                  te_V.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dK.data(),
                  te_dV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dK, dV, dBias};
}

namespace flash_attention {

constexpr int warp_size = 32;
constexpr int type_size = 2;  // FP16 or BF16
constexpr int nvec = sizeof(uint64_t) / type_size;
constexpr int load_size = warp_size * nvec;
constexpr int block_size = 512;

template <typename T>
__launch_bounds__(block_size)
__global__ void prepare_kernel_fwd(const T *qkvi,
                                   T *qkv,
                                   const size_t B,
                                   const size_t S,
                                   const size_t Z,
                                   const size_t W) {
    const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
    const int id_in_warp = threadIdx.x % warp_size;
    const size_t offset_input = blockIdx.y * W + warpid * 3 * W * Z + id_in_warp * nvec;
    const T *my_input = qkvi + offset_input;

    const size_t s = warpid / B;
    if (s >= S) return;

    const size_t b = warpid % B;

    const size_t offset_output = blockIdx.y * B * S * Z * W +
                                 (s + b * S) * W * Z +
                                 id_in_warp * nvec;

    T *my_output = qkv + offset_output;

    for (int i = 0; i < Z; ++i) {
        uint64_t *out = reinterpret_cast<uint64_t*>(my_output + i * load_size);
        *out = *reinterpret_cast<const uint64_t*>(my_input + i * load_size * 3);
    }
}

template <typename T>
__launch_bounds__(block_size)
__global__ void prepare_kernel_bwd(const T *q, const T *k, const T *v,
                                   T *qkv, const size_t B, const size_t S,
                                   const size_t Z, const size_t W) {
    const T *input = blockIdx.y == 0 ? q : (blockIdx.y == 1 ? k : v);

    const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
    const int id_in_warp = threadIdx.x % warp_size;
    const size_t offset_input = warpid * W * Z + id_in_warp * nvec;
    const T *my_input = input + offset_input;

    const size_t b = warpid / S;
    if (b >= B) return;

    const size_t s = warpid % S;

    const size_t offset_output = (b + s * B) * 3 * W * Z +
                                 id_in_warp * nvec + blockIdx.y * W;

    T *my_output = qkv + offset_output;

    for (int i = 0; i < Z; ++i) {
        uint64_t *out = reinterpret_cast<uint64_t*>(my_output + i * load_size * 3);
        *out = *reinterpret_cast<const uint64_t*>(my_input + i * load_size);
    }
}

}  // namespace flash_attention

at::Tensor fa_prepare_fwd(at::Tensor qkvi) {
    NVTE_CHECK(qkvi.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(qkvi.scalar_type() == at::ScalarType::Half ||
               qkvi.scalar_type() == at::ScalarType::BFloat16);
    NVTE_CHECK(qkvi.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(qkvi.size(3) == flash_attention::load_size);
    NVTE_CHECK(qkvi.stride(3) == 1, "Wrong stride.");
    NVTE_CHECK(qkvi.stride(2) == 3 * qkvi.size(3), "Wrong stride.");
    NVTE_CHECK(qkvi.stride(1) == 3 * qkvi.size(3) * qkvi.size(2), "Wrong stride.");
    NVTE_CHECK(qkvi.stride(0) == 3 * qkvi.size(3) * qkvi.size(2) * qkvi.size(1), "Wrong stride.");

    // [s, b, n, h * 3] -> [3, b, s, n, h]
    std::vector<int64_t> shape = {3, qkvi.size(1), qkvi.size(0), qkvi.size(2), qkvi.size(3)};
    at::Tensor qkv = at::empty(shape, at::CUDA(qkvi.scalar_type()));

    size_t warps = qkvi.size(0) * qkvi.size(1);
    size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
    size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
    dim3 grid(blocks, 3);
    int threads = flash_attention::block_size;
    if (qkvi.scalar_type() == at::ScalarType::Half) {
        using dtype = at::Half;
        flash_attention::prepare_kernel_fwd<dtype><<<grid, threads, 0,
                                                     at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            shape[1],
            shape[2],
            shape[3],
            shape[4]);
    } else {
        using dtype = at::BFloat16;
        flash_attention::prepare_kernel_fwd<dtype><<<grid, threads, 0,
                                                     at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            shape[1],
            shape[2],
            shape[3],
            shape[4]);
    }

    return qkv;
}

at::Tensor fa_prepare_bwd(at::Tensor q, at::Tensor k, at::Tensor v) {
    NVTE_CHECK(q.is_contiguous());
    NVTE_CHECK(k.is_contiguous());
    NVTE_CHECK(v.is_contiguous());
    NVTE_CHECK(q.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(k.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(v.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(q.scalar_type() == at::ScalarType::Half ||
               q.scalar_type() == at::ScalarType::BFloat16);
    NVTE_CHECK(k.scalar_type() == q.scalar_type());
    NVTE_CHECK(v.scalar_type() == q.scalar_type());
    NVTE_CHECK(q.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(q.size(3) == flash_attention::load_size);
    NVTE_CHECK(k.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(k.size(3) == flash_attention::load_size);
    NVTE_CHECK(v.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(v.size(3) == flash_attention::load_size);

    // 3 x [s, b, n, h] -> [b, s, n, 3 * h]

    std::vector<int64_t> shape = {q.size(1), q.size(0), q.size(2), 3 * q.size(3)};
    at::Tensor qkv = at::empty(shape, at::CUDA(q.scalar_type()));

    size_t warps = q.size(0) * q.size(1);
    size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
    size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
    dim3 grid(blocks, 3);
    int threads = flash_attention::block_size;
    if (q.scalar_type() == at::ScalarType::Half) {
        using dtype = at::Half;
        flash_attention::prepare_kernel_bwd<dtype><<<grid, threads, 0,
                                                 at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(),
            k.data_ptr<dtype>(),
            v.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            q.size(0),
            q.size(1),
            q.size(2),
            q.size(3));
    } else {
        using dtype = at::BFloat16;
        flash_attention::prepare_kernel_bwd<dtype><<<grid, threads, 0,
                                                 at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(),
            k.data_ptr<dtype>(),
            v.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            q.size(0),
            q.size(1),
            q.size(2),
            q.size(3));
    }

    return qkv;
}

/***************************************************************************************************
 * Support THD format for Context Parallel
 **************************************************************************************************/

__forceinline__
__device__ int binary_search(int target, int *array, int len) {
  int left = 1, right = len - 1;
  while (left < right) {
    int mid = (left + right) / 2;
    if (array[mid] <= target) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }
  return left - 1;
}

template <typename T, typename Functor>
__global__ void thd_segment_elementwise_kernel(T *t1,
                                               const T *t2,
                                               const int *offset1,
                                               const int *offset2,
                                               const int *cu_lens,
                                               int batch,
                                               int hidden_size,
                                               int len1,
                                               int len2) {
  extern __shared__ int s_cu_lens[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    s_cu_lens[i] = cu_lens[i];
  }
  __syncthreads();

  const int warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  const int lane_id = threadIdx.x % 32;
  const int num_warps = blockDim.x * gridDim.x / 32;
  const int total_tokens = s_cu_lens[batch];
  const int loops_per_token = hidden_size * sizeof(T) / sizeof(int4);

  t1 += (size_t)blockIdx.y * hidden_size * len1;
  t2 += (size_t)blockIdx.y * hidden_size * len2;

  for (int i = warp_id; i < total_tokens; i += num_warps) {
    int seq_id = binary_search(i, s_cu_lens, batch + 1);

    size_t idx1 = i - s_cu_lens[seq_id] + offset1[seq_id];
    size_t idx2 = i - s_cu_lens[seq_id] + offset2[seq_id];
    int4 *p1 = (int4*)&t1[idx1 * hidden_size];
    int4 *p2 = (int4*)&t2[idx2 * hidden_size];

    for (int j = lane_id; j < loops_per_token; j += 32) {
      Functor::execute(&p1[j], &p2[j]);
    }
  }
}

template <typename T, typename Functor>
__global__ void thd_segment_lse_kernel(T *lse1,
                                       const float *lse2,
                                       const int *offset1,
                                       const int *offset2,
                                       const int *cu_lens,
                                       int batch,
                                       int num_heads,
                                       int max_len1,
                                       int max_len2) {
  extern __shared__ int s_cu_lens[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    s_cu_lens[i] = cu_lens[i];
  }
  __syncthreads();

  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int num_threads = blockDim.x * gridDim.x;
  const size_t total_heads = (size_t)s_cu_lens[batch] * num_heads;

  for (size_t i = tid; i < total_heads; i += num_threads) {
    int head_id  = i % num_heads;
    int token_id = i / num_heads;
    int seq_id   = binary_search(token_id, s_cu_lens, batch + 1);

    int token_in_seq = token_id - s_cu_lens[seq_id];
    size_t row_offset = (size_t)seq_id * num_heads + head_id;
    size_t idx1 = token_in_seq + offset1[seq_id] + row_offset * max_len1;
    size_t idx2 = token_in_seq + offset2[seq_id] + row_offset * max_len2;

    Functor::execute(&lse1[idx1], &lse2[idx2]);
  }
}

template <typename T, int tile_size>
__global__ void thd_out_correction_kernel(T *out1,
                                          const T *out2,
                                          const float *lse1,
                                          const float *lse2,
                                          const int *out_offset1,
                                          const int *out_offset2,
                                          const int *lse_offset1,
                                          const int *lse_offset2,
                                          const int *cu_lens,
                                          int batch,
                                          int num_heads,
                                          int head_dim,
                                          int max_len1,
                                          int max_len2) {
  extern __shared__ int s_cu_lens[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    s_cu_lens[i] = cu_lens[i];
  }
  __syncthreads();

  const int tile_id = (blockIdx.x * blockDim.x + threadIdx.x) / tile_size;
  const int lane_id = threadIdx.x % tile_size;
  const int num_tiles = blockDim.x * gridDim.x / tile_size;
  const size_t total_heads = (size_t)s_cu_lens[batch] * num_heads;
  const int loops_per_head = head_dim * sizeof(T) / sizeof(int4);

  for (size_t i = tile_id; i < total_heads; i += num_tiles) {
    int head_id  = i % num_heads;
    int token_id = i / num_heads;
    int seq_id   = binary_search(token_id, s_cu_lens, batch + 1);

    int token_in_seq = token_id - s_cu_lens[seq_id];
    size_t idx1, idx2;

    // Get the corrected softmax_lse
    size_t lse_row_offset = (size_t)seq_id * num_heads + head_id;
    idx1 = token_in_seq + lse_offset1[seq_id] + lse_row_offset * max_len1;
    idx2 = token_in_seq + lse_offset2[seq_id] + lse_row_offset * max_len2;
    float lse = exp(lse2[idx2] - lse1[idx1]);

    // Get the pointers of heads from out1 and out2
    idx1 = ((size_t)(token_in_seq + out_offset1[seq_id]) * num_heads + head_id) * head_dim;
    idx2 = ((size_t)(token_in_seq + out_offset2[seq_id]) * num_heads + head_id) * head_dim;
    int4 *p1 = (int4*)&out1[idx1];
    int4 *p2 = (int4*)&out2[idx2];

    // Do correction for a head
    for (int j = lane_id; j < loops_per_head; j += tile_size) {
      int4 storage1 = p1[j];
      int4 storage2 = p2[j];
      T *s1 = (T*)&storage1;
      T *s2 = (T*)&storage2;

      #pragma unroll
      for (int k = 0; k < sizeof(int4) / sizeof(T); k++) {
        s1[k] += s2[k] * lse;
      }

      p1[j] = storage1;
    }
  }
}

template <typename T>
struct NewAddFunctor {
  __forceinline__ __device__
  static void execute(int4 *p1, const int4 *p2) {
    int4 storage1 = *p1;
    int4 storage2 = *p2;
    T *s1 = (T*)&storage1;
    T *s2 = (T*)&storage2;

    #pragma unroll
    for (int i = 0; i < sizeof(int4) / sizeof(T); i++) {
      s1[i] += s2[i];
    }

    *p1 = storage1;
  }
};

template <typename T>
struct NewCopyFunctor {
  __forceinline__ __device__
  static void execute(T *p1, const T *p2) {
    *p1 = *p2;
  }
};

struct NewLseFunctor {
  __forceinline__ __device__
  static void execute(double *lse, const float *lse_per_step) {
    double val = *lse;
    double val_per_step = (double)(*lse_per_step);
    double max_scale = max(val, val_per_step);
    double min_scale = min(val, val_per_step);
    *lse = max_scale + log(1.0 + exp(min_scale - max_scale));
  }
};

template <typename T, typename Functor>
void thd_segment_elementwise_helper(at::Tensor &t1,
                                    const at::Tensor &t2,
                                    const at::Tensor &offset1,
                                    const at::Tensor &offset2,
                                    const at::Tensor &cu_lens,
                                    size_t total_len) {
  NVTE_CHECK(t1.dim() == 3 || t1.dim() == 4);
  NVTE_CHECK(t2.dim() == t1.dim());
  NVTE_CHECK(t1.scalar_type() == t2.scalar_type());

  NVTE_CHECK(t1.size(t1.dim() - 1) == t2.size(t2.dim() - 1));
  NVTE_CHECK(t1.size(t1.dim() - 2) == t2.size(t2.dim() - 2));
  if (t1.dim() == 4) {
    NVTE_CHECK(t1.size(0) == t2.size(0));
  }

  NVTE_CHECK(offset1.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(offset2.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_lens.scalar_type() == at::ScalarType::Int);

  NVTE_CHECK(offset1.dim() == 1);
  NVTE_CHECK(offset2.dim() == 1);
  NVTE_CHECK(cu_lens.dim() == 1);

  NVTE_CHECK(cu_lens.size(0) >= 2);
  NVTE_CHECK(offset1.size(0) == cu_lens.size(0) || offset1.size(0) == cu_lens.size(0) - 1);
  NVTE_CHECK(offset2.size(0) == cu_lens.size(0) || offset2.size(0) == cu_lens.size(0) - 1);

  int batch = cu_lens.size(0) - 1;
  int seq_dim = t1.dim() == 3 ? 0 : 1;
  int hidden_size = t1.size(seq_dim + 1) * t1.size(seq_dim + 2);

  // For 128-bits load/store
  NVTE_CHECK((hidden_size * sizeof(T)) % 16 == 0);

  unsigned int block = 256;
  unsigned int num_blocks = (total_len * 32 + block - 1) / block;
  dim3 grid(num_blocks, seq_dim + 1);
  thd_segment_elementwise_kernel<T, Functor><<<grid, block, sizeof(int) * (batch + 1),
                                               at::cuda::getCurrentCUDAStream()>>>(
    t1.data_ptr<T>(),
    t2.data_ptr<T>(),
    offset1.data_ptr<int>(),
    offset2.data_ptr<int>(),
    cu_lens.data_ptr<int>(),
    batch,
    hidden_size,
    t1.size(seq_dim),
    t2.size(seq_dim));
}

template <typename T, typename Functor>
void thd_segment_lse_helper(at::Tensor &lse1,
                            const at::Tensor &lse2,
                            const at::Tensor &offset1,
                            const at::Tensor &offset2,
                            const at::Tensor &cu_lens,
                            size_t total_len) {
  NVTE_CHECK(lse1.dim() == 3);
  NVTE_CHECK(lse2.dim() == 3);
  NVTE_CHECK(lse1.size(0) == lse2.size(0));
  NVTE_CHECK(lse1.size(1) == lse2.size(1));

  NVTE_CHECK(offset1.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(offset2.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_lens.scalar_type() == at::ScalarType::Int);

  NVTE_CHECK(offset1.dim() == 1);
  NVTE_CHECK(offset2.dim() == 1);
  NVTE_CHECK(cu_lens.dim() == 1);

  NVTE_CHECK(cu_lens.size(0) >= 2);
  NVTE_CHECK(cu_lens.size(0) == lse1.size(0) + 1);
  NVTE_CHECK(offset1.size(0) == cu_lens.size(0) || offset1.size(0) == cu_lens.size(0) - 1);
  NVTE_CHECK(offset2.size(0) == cu_lens.size(0) || offset2.size(0) == cu_lens.size(0) - 1);

  int batch     = lse1.size(0);
  int num_heads = lse1.size(1);
  int max_len1  = lse1.size(2);
  int max_len2  = lse2.size(2);

  unsigned int block = 256;
  unsigned int num_blocks = (total_len * num_heads + block - 1) / block;
  dim3 grid(num_blocks);
  thd_segment_lse_kernel<T, Functor><<<grid, block, sizeof(int) * (batch + 1),
                                       at::cuda::getCurrentCUDAStream()>>>(
    lse1.data_ptr<T>(),
    lse2.data_ptr<float>(),
    offset1.data_ptr<int>(),
    offset2.data_ptr<int>(),
    cu_lens.data_ptr<int>(),
    batch,
    num_heads,
    max_len1,
    max_len2);
}

template <typename T>
void thd_seg_out_correction_helper(at::Tensor &out1,
                                   const at::Tensor &out2,
                                   const at::Tensor &lse1,
                                   const at::Tensor &lse2,
                                   const at::Tensor &out_offset1,
                                   const at::Tensor &out_offset2,
                                   const at::Tensor &lse_offset1,
                                   const at::Tensor &lse_offset2,
                                   const at::Tensor &cu_lens,
                                   size_t total_len) {
  NVTE_CHECK(out1.device().is_cuda());

  NVTE_CHECK(out1.dim() == 3 && out2.dim() == 3);
  NVTE_CHECK(lse1.dim() == 3 && lse2.dim() == 3);
  NVTE_CHECK(out_offset1.dim() == 1 && out_offset2.dim() == 1);
  NVTE_CHECK(lse_offset1.dim() == 1 && lse_offset2.dim() == 1);
  NVTE_CHECK(cu_lens.dim() == 1);

  NVTE_CHECK(out1.size(1) == out2.size(1));
  NVTE_CHECK(out1.size(2) == out2.size(2));
  NVTE_CHECK(lse1.size(0) == lse2.size(0));
  NVTE_CHECK(lse1.size(1) == lse2.size(1));

  // NVTE_CHECK(out_offset1.size(0) >= 2);
  // NVTE_CHECK(out_offset1.size(0) == out_offset2.size(0));
  // NVTE_CHECK(out_offset1.size(0) == lse_offset1.size(0));
  // NVTE_CHECK(out_offset1.size(0) == lse_offset2.size(0));

  NVTE_CHECK(cu_lens.size(0) >= 2);
  NVTE_CHECK(cu_lens.size(0) == lse1.size(0) + 1);

  NVTE_CHECK(out_offset1.size(0) == cu_lens.size(0) || out_offset1.size(0) == cu_lens.size(0) - 1);
  NVTE_CHECK(out_offset2.size(0) == cu_lens.size(0) || out_offset2.size(0) == cu_lens.size(0) - 1);
  NVTE_CHECK(lse_offset1.size(0) == cu_lens.size(0) || lse_offset1.size(0) == cu_lens.size(0) - 1);
  NVTE_CHECK(lse_offset2.size(0) == cu_lens.size(0) || lse_offset2.size(0) == cu_lens.size(0) - 1);

  int batch     = lse1.size(0);
  int num_heads = lse1.size(1);
  int max_len1  = lse1.size(2);
  int max_len2  = lse2.size(2);
  int head_dim  = out1.size(2);

  constexpr int tile_size = 16;
  unsigned int block = 256;
  unsigned int num_blocks = (total_len * tile_size + block - 1) / block;
  dim3 grid(num_blocks);
  thd_out_correction_kernel<T, tile_size><<<grid, block, sizeof(int) * (batch + 1),
                                            at::cuda::getCurrentCUDAStream()>>>(
    out1.data_ptr<T>(),
    out2.data_ptr<T>(),
    lse1.data_ptr<float>(),
    lse2.data_ptr<float>(),
    out_offset1.data_ptr<int>(),
    out_offset2.data_ptr<int>(),
    lse_offset1.data_ptr<int>(),
    lse_offset2.data_ptr<int>(),
    cu_lens.data_ptr<int>(),
    batch,
    num_heads,
    head_dim,
    max_len1,
    max_len2);
}

void thd_segment_add(at::Tensor &t1,
                     const at::Tensor &t2,
                     const at::Tensor &offset1,
                     const at::Tensor &offset2,
                     const at::Tensor &cu_lens,
                     size_t total_len) {
  if (t1.scalar_type() == at::ScalarType::Half) {
    using T = at::Half;
    thd_segment_elementwise_helper<T, NewAddFunctor<T>>(t1, t2, offset1, offset2, cu_lens, total_len);
  } else if (t1.scalar_type() == at::ScalarType::BFloat16) {
    using T = at::BFloat16;
    thd_segment_elementwise_helper<T, NewAddFunctor<T>>(t1, t2, offset1, offset2, cu_lens, total_len);
  } else if (t1.scalar_type() == at::ScalarType::Float) {
    using T = float;
    thd_segment_elementwise_helper<T, NewAddFunctor<T>>(t1, t2, offset1, offset2, cu_lens, total_len);
  } else {
    NVTE_ERROR("thd_segment_add only supports FP16/BF16/FP32 data types. \n");
  }
}

void thd_segment_copy(at::Tensor &t1,
                      const at::Tensor &t2,
                      const at::Tensor &offset1,
                      const at::Tensor &offset2,
                      const at::Tensor &cu_lens,
                      size_t total_len) {
  if (t1.scalar_type() == at::ScalarType::Half) {
    using T = at::Half;
    thd_segment_elementwise_helper<T, NewCopyFunctor<int4>>(t1, t2, offset1, offset2, cu_lens, total_len);
  } else if (t1.scalar_type() == at::ScalarType::BFloat16) {
    using T = at::BFloat16;
    thd_segment_elementwise_helper<T, NewCopyFunctor<int4>>(t1, t2, offset1, offset2, cu_lens, total_len);
  } else if (t1.scalar_type() == at::ScalarType::Float) {
    using T = float;
    thd_segment_elementwise_helper<T, NewCopyFunctor<int4>>(t1, t2, offset1, offset2, cu_lens, total_len);
  } else {
    NVTE_ERROR("thd_segment_copy only supports FP16/BF16/FP32 data types. \n");
  }
}

void thd_seg_lse_correction(at::Tensor &lse1, const at::Tensor &lse2,
                            const at::Tensor &offset1, const at::Tensor &offset2,
                            const at::Tensor &cu_lens, size_t total_len) {
  thd_segment_lse_helper<double, NewLseFunctor>(
    lse1, lse2, offset1, offset2, cu_lens, total_len);
}

void thd_seg_read_lse(at::Tensor &lse1, const at::Tensor &lse2,
                      const at::Tensor &offset1, const at::Tensor &offset2,
                      const at::Tensor &cu_lens, size_t total_len) {
  thd_segment_lse_helper<float, NewCopyFunctor<float>>(
    lse1, lse2, offset1, offset2, cu_lens, total_len);
}

void thd_seg_out_correction(at::Tensor &out1, const at::Tensor &out2, const at::Tensor &lse1,
                            const at::Tensor &lse2, const at::Tensor &out_offset1,
                            const at::Tensor &out_offset2, const at::Tensor &lse_offset1,
                            const at::Tensor &lse_offset2, const at::Tensor &cu_lens,
                            size_t total_len) {
  if (out1.scalar_type() == at::ScalarType::Half) {
    thd_seg_out_correction_helper<at::Half>(
      out1, out2, lse1, lse2, out_offset1, out_offset2, lse_offset1, lse_offset2, cu_lens,
      total_len);
  } else if (out1.scalar_type() == at::ScalarType::BFloat16) {
    thd_seg_out_correction_helper<at::BFloat16>(
      out1, out2, lse1, lse2, out_offset1, out_offset2, lse_offset1, lse_offset2, cu_lens,
      total_len);
  } else if (out1.scalar_type() == at::ScalarType::Float) {
    thd_seg_out_correction_helper<float>(
      out1, out2, lse1, lse2, out_offset1, out_offset2, lse_offset1, lse_offset2, cu_lens,
      total_len);
  } else {
    NVTE_ERROR("thd_seg_out_correction only supports FP16/BF16/FP32 data types. \n");
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Read the half of a THD tensor
 **************************************************************************************************/

__global__ void thd_read_half_tensor_kernel(void *half,
                                            void *tensor,
                                            int *cu_seqlens,
                                            int batch,
                                            int hidden_size_in_bytes,
                                            int half_idx,
                                            int token_offset) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int warpid = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  int laneid = threadIdx.x % 32;
  int num_warps = (blockDim.x * gridDim.x) / 32;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_float4s_per_token = hidden_size_in_bytes / sizeof(float4);

  size_t offset = token_offset * (size_t)hidden_size_in_bytes;
  half = (void*)((char*)half + offset/2 * blockIdx.y);
  tensor = (void*)((char*)tensor + offset * blockIdx.y);

  for (int token_id = warpid; token_id < num_total_tokens; token_id += num_warps) {
    int seqid = binary_search(token_id, cu_seqlens_s, batch + 1);

    size_t offset_in_bytes = (size_t)token_id * hidden_size_in_bytes;
    float4* cur_half_token = (float4*)((char*)half + offset_in_bytes);

    offset_in_bytes = ((size_t)token_id + cu_seqlens_s[seqid + half_idx]) * hidden_size_in_bytes;
    float4* cur_token = (float4*)((char*)tensor + offset_in_bytes);

    for (int idx = laneid; idx < num_float4s_per_token; idx += 32) {
      cur_half_token[idx] = cur_token[idx];
    }
  }
}

at::Tensor thd_read_half_tensor(const at::Tensor &tensor,
                                const at::Tensor &cu_seqlens,
                                int half_idx) {
  NVTE_CHECK(tensor.dim() == 3 || tensor.dim() == 4);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);

  // Shapes of q and dq are [t, h, d], so the dimension of "t" is 0
  // Shapes of kv and dkv are [2, t, h, d], so the dimension of "t" is 1
  int seq_dim = tensor.dim() == 3 ? 0 : 1;

  int batch = cu_seqlens.size(0) - 1;
  int num_heads    = tensor.size(seq_dim + 1);
  int dim_per_head = tensor.size(seq_dim + 2);
  size_t hidden_size_in_bytes = num_heads * dim_per_head * c10::elementSize(tensor.scalar_type());

  NVTE_CHECK(hidden_size_in_bytes % 16 == 0);

  // Generate output
  std::vector<int64_t> shape(tensor.dim());
  for (size_t i = 0; i < shape.size(); i++) {
    shape[i] = tensor.size(i);
  }
  shape[seq_dim] /= 2;
  at::Tensor half = at::empty(shape, at::CUDA(tensor.scalar_type()));

  // Launch Kernel
  constexpr unsigned int block = 256;
  unsigned int grid_x = (tensor.size(seq_dim) / 2 * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= tensor.size(i);
  }
  dim3 grid = {grid_x, grid_y};
  thd_read_half_tensor_kernel<<<grid, block, sizeof(int) * (batch+1),
                                at::cuda::getCurrentCUDAStream()>>>(
    half.data_ptr(),
    tensor.data_ptr(),
    cu_seqlens.data_ptr<int>(),
    batch,
    hidden_size_in_bytes,
    half_idx,
    tensor.size(seq_dim));

  return half;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: softmax_lse related operations
 **************************************************************************************************/

template <typename lse_dtype, typename Functor>
__global__ void thd_lse_kernel(lse_dtype *lse, float *half_lse, int *cu_seqlens,
                               int batch, int num_heads, int max_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  int num_total_tokens = cu_seqlens_s[batch];

  for (int token_id = tid; token_id < num_total_tokens; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      int row = seq_id * num_heads + head_id;
      int col = token_id - cu_seqlens_s[seq_id];
      int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];

      size_t idx = (size_t)row * max_seqlen + col + seq_len;
      size_t half_idx = (size_t)row * max_seqlen / 2 + col;

      Functor::run(lse, half_lse, idx, half_idx);
    }
  }
}

struct LseCorrectionFunctor {
  __forceinline__
  __device__ static void run(double *lse, float *half_lse, size_t idx, size_t half_idx) {
    double val = lse[idx];
    float val_per_step = half_lse[half_idx];
    double max_scale = max(val, val_per_step);
    double min_scale = min(val, val_per_step);
    lse[idx] = max_scale + log(1.0 + exp(min_scale - max_scale));
  }
};

void thd_lse_correction(at::Tensor &lse,
                        const at::Tensor &lse_per_step,
                        const at::Tensor &cu_seqlens,
                        int total_tokens) {
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Double);
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);

  NVTE_CHECK(lse.dim() == 3);
  NVTE_CHECK(lse_per_step.dim() == 3);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch      = lse.size(0);
  int num_heads  = lse.size(1);
  int max_seqlen = lse.size(2);

  NVTE_CHECK(lse_per_step.size(0) == batch);
  NVTE_CHECK(lse_per_step.size(1) == num_heads);
  NVTE_CHECK(lse_per_step.size(2) == max_seqlen / 2);
  NVTE_CHECK(cu_seqlens.size(0) == batch + 1);

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};
  thd_lse_kernel<double, LseCorrectionFunctor><<<grid, block, sizeof(int) * (batch+1),
                                                 at::cuda::getCurrentCUDAStream()>>>(
    lse.data_ptr<double>(),
    lse_per_step.data_ptr<float>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    num_heads,
    max_seqlen);
}

struct ReadLseFunctor {
  __forceinline__
  __device__ static void run(float *lse, float *half_lse, size_t idx, size_t half_idx) {
    half_lse[half_idx] = lse[idx];
  }
};

at::Tensor thd_read_half_lse(const at::Tensor &lse,
                             const at::Tensor &cu_seqlens,
                             int total_tokens) {
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(lse.dim() == 3);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch      = lse.size(0);
  int num_heads  = lse.size(1);
  int max_seqlen = lse.size(2);

  NVTE_CHECK(cu_seqlens.size(0) == batch + 1);

  std::vector<int64_t> shape = {batch, num_heads, max_seqlen / 2};
  at::Tensor half_lse = at::zeros(shape, at::CUDA(lse.scalar_type()));

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};
  thd_lse_kernel<float, ReadLseFunctor><<<grid, block, sizeof(int) * (batch+1),
                                          at::cuda::getCurrentCUDAStream()>>>(
    lse.data_ptr<float>(),
    half_lse.data_ptr<float>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    num_heads,
    max_seqlen);

  return half_lse;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Out correction in forward
 **************************************************************************************************/

template <typename dtype, int is_half, int tile_size>
__global__ void thd_out_correction_kernel(dtype *out,
                                          dtype *out_per_step,
                                          float *lse,
                                          float *lse_per_step,
                                          int *cu_seqlens,
                                          int batch,
                                          int num_heads,
                                          int dim_per_head,
                                          int max_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / (is_half + 1);
  }
  __syncthreads();

  int tile_id = (blockIdx.x * blockDim.x + threadIdx.x) / tile_size;
  int lane_id = threadIdx.x % tile_size;
  int num_tiles = (blockDim.x * gridDim.x) / tile_size;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_loops_per_head = dim_per_head * sizeof(dtype) / sizeof(float4);

  for (int token_id = tile_id; token_id < num_total_tokens; token_id += num_tiles) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      size_t idx, idx_per_step;

      int row = seq_id * num_heads + head_id;
      int col = token_id - cu_seqlens_s[seq_id];
      int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
      idx = (size_t)row * max_seqlen + col + seq_len * is_half;
      idx_per_step = (size_t)row * max_seqlen / (is_half + 1) + col;
      float lse_corrected_exp = exp(lse_per_step[idx_per_step] - lse[idx]);

      idx = (size_t)token_id + cu_seqlens_s[seq_id + 1] * is_half;
      idx = (idx * num_heads + head_id) * dim_per_head;
      idx_per_step = ((size_t)token_id * num_heads + head_id) * dim_per_head;
      dtype *cur_out = out + idx;
      dtype *cur_out_per_step = out_per_step + idx_per_step;

      for (int j = lane_id; j < num_loops_per_head; j += tile_size) {
        float4 data_per_step = ((float4*)cur_out_per_step)[j];
        float4 data          = ((float4*)cur_out)[j];
        dtype *p_per_step = (dtype*)&data_per_step;
        dtype *p          = (dtype*)&data;
        for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
          p[i] += p_per_step[i] * lse_corrected_exp;
        }
        ((float4*)cur_out)[j] = data;
      }
    }
  }
}

template<typename dtype, int is_half>
static void thd_out_correction_helper(at::Tensor &out,
                                      const at::Tensor &out_per_step,
                                      const at::Tensor &lse,
                                      const at::Tensor &lse_per_step,
                                      const at::Tensor &cu_seqlens) {
  NVTE_CHECK(out.scalar_type() == out_per_step.scalar_type());
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);

  int total_tokens = out.size(0);
  int num_heads    = out.size(1);
  int dim_per_head = out.size(2);
  int batch      = lse.size(0);
  int max_seqlen = lse.size(2);

  NVTE_CHECK(out_per_step.size(0) == total_tokens / (is_half + 1));
  NVTE_CHECK(out_per_step.size(1) == num_heads);
  NVTE_CHECK(out_per_step.size(2) == dim_per_head);
  NVTE_CHECK(lse.size(1) == num_heads);
  NVTE_CHECK(lse_per_step.size(0) == batch);
  NVTE_CHECK(lse_per_step.size(1) == num_heads);
  NVTE_CHECK(lse_per_step.size(2) == max_seqlen / (is_half + 1));
  NVTE_CHECK(cu_seqlens.size(0) == batch + 1);

  constexpr int tile = 16;
  constexpr int block = 512;
  unsigned int grid_x = min((total_tokens / (is_half + 1) * tile + block - 1) / block, 256);
  dim3 grid = {grid_x, (unsigned int)num_heads};

  thd_out_correction_kernel<dtype, is_half, tile><<<grid, block, sizeof(int) * (batch+1),
                                                    at::cuda::getCurrentCUDAStream()>>>(
    out.data_ptr<dtype>(),
    out_per_step.data_ptr<dtype>(),
    lse.data_ptr<float>(),
    lse_per_step.data_ptr<float>(),
    cu_seqlens.data_ptr<int>(),
    batch,
    num_heads,
    dim_per_head,
    max_seqlen);
}

void thd_out_correction(at::Tensor &out,
                        const at::Tensor &out_per_step,
                        const at::Tensor &lse,
                        const at::Tensor &lse_per_step,
                        const at::Tensor &cu_seqlens,
                        bool is_half) {
  if (is_half) {
    if (out.scalar_type() == at::ScalarType::Half) {
      using dtype = at::Half;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens);
    } else if (out.scalar_type() == at::ScalarType::BFloat16) {
      using dtype = at::BFloat16;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens);
    } else if (out.scalar_type() == at::ScalarType::Float) {
      using dtype = float;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens);
    } else {
      NVTE_ERROR("Unsupported dtype of out\n");
    }
  } else {
    if (out.scalar_type() == at::ScalarType::Half) {
      using dtype = at::Half;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens);
    } else if (out.scalar_type() == at::ScalarType::BFloat16) {
      using dtype = at::BFloat16;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens);
    } else if (out.scalar_type() == at::ScalarType::Float) {
      using dtype = float;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens);
    } else {
      NVTE_ERROR("Unsupported dtype of out\n");
    }
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Gradients correction in backward
 **************************************************************************************************/

template <typename dtype, typename Functor_0, typename Functor_1, int functor_idx, int group_size>
__global__ void thd_grad_correction_kernel(dtype *grad, dtype *grad_per_step, int *cu_seqlens,
                                           int batch, int hidden_size, int token_offset) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    if constexpr (functor_idx < 2) {
      cu_seqlens_s[i] = cu_seqlens[i] / 2;
    } else {
      cu_seqlens_s[i] = cu_seqlens[i];
    }
  }
  __syncthreads();

  int group_id = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int lane_id = threadIdx.x % group_size;
  int num_groups = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_inner_loops = hidden_size * sizeof(dtype) / sizeof(float4);

  size_t offset = token_offset * (size_t)hidden_size;
  if constexpr (functor_idx < 2) {
    grad_per_step = grad_per_step + offset / 2 * blockIdx.y;
  } else {
    grad_per_step = grad_per_step + offset * blockIdx.y;
  }
  grad = grad + offset * blockIdx.y;

  for (int token_id = group_id; token_id < num_total_tokens; token_id += num_groups) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);

    int token_offset;
    bool is_first_half;
    if constexpr (functor_idx < 2) {
      token_offset = cu_seqlens_s[seq_id + functor_idx];
      is_first_half = (functor_idx == 0);
    } else {
      token_offset = 0;
      int len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
      is_first_half = (token_id - cu_seqlens_s[seq_id]) < (len / 2);
    }

    dtype *token = &grad[(token_id + token_offset) * (size_t)hidden_size];
    dtype *token_per_step = &grad_per_step[token_id * (size_t)hidden_size];
    for (int idx = lane_id; idx < num_inner_loops; idx += group_size) {
      if (is_first_half) {
        Functor_0::run(token, token_per_step, idx);
      } else {
        Functor_1::run(token, token_per_step, idx);
      }
    }
  }
}

struct EmptyFunctor {
  __forceinline__
  __device__ static void run(void *token, void *token_per_step, int idx) {
  }
};

struct CopyFunctor {
  __forceinline__
  __device__ static void run(void *token, void *token_per_step, int idx) {
    ((float4*)token)[idx] = ((float4*)token_per_step)[idx];
  }
};

template <typename dtype>
struct AddFunctor {
  __forceinline__
  __device__ static void run(dtype *token, dtype *token_per_step, int idx) {
    float4 d = ((float4*)token)[idx];
    dtype *p = (dtype*)(&d);
    float4 d_ = ((float4*)token_per_step)[idx];
    dtype *p_ = (dtype*)(&d_);
    for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
      p[i] += p_[i];
    }
    ((float4*)token)[idx] = d;
  }
};

template <typename dtype, typename Functor_0, typename Functor_1, int functor_idx>
static void thd_grad_correction_helper(at::Tensor &grad,
                                       const at::Tensor &grad_per_step,
                                       const at::Tensor &cu_seqlens) {
  NVTE_CHECK(grad.dim() == 3 || grad.dim() == 4);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  // Shape of dq is [t, h, d], so the dimension of "t" is 0
  // Shape of dkv is [2, t, h, d], so the dimension of "t" is 1
  int seq_dim = grad.dim() == 3 ? 0 : 1;

  int total_tokens = grad.size(seq_dim);
  int num_heads    = grad.size(seq_dim + 1);
  int dim_per_head = grad.size(seq_dim + 2);
  int batch = cu_seqlens.size(0) - 1;

  if constexpr (functor_idx < 2) {
    NVTE_CHECK(grad_per_step.size(seq_dim) == total_tokens / 2);
  } else {
    NVTE_CHECK(grad_per_step.size(seq_dim) == total_tokens);
  }
  NVTE_CHECK(grad_per_step.size(seq_dim + 1) == num_heads);
  NVTE_CHECK(grad_per_step.size(seq_dim + 2) == dim_per_head);

  size_t hidden_size = num_heads * dim_per_head;
  NVTE_CHECK((hidden_size * c10::elementSize(grad.scalar_type())) % 16 == 0);

  constexpr unsigned int block = 256;
  unsigned int grid_x;
  if constexpr (functor_idx < 2) {
    grid_x = (total_tokens / 2 * 32 + block - 1) / block;
  } else {
    grid_x = (total_tokens * 32 + block - 1) / block;
  }
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= grad.size(i);
  }
  dim3 grid = {grid_x, grid_y};

  thd_grad_correction_kernel<dtype, Functor_0, Functor_1, functor_idx, 32>
    <<<grid, block, sizeof(int) * (batch+1), at::cuda::getCurrentCUDAStream()>>>(
      grad.data_ptr<dtype>(),
      grad_per_step.data_ptr<dtype>(),
      cu_seqlens.data_ptr<int>(),
      batch,
      hidden_size,
      total_tokens);
}

template <typename dtype>
static void thd_grad_dispatcher(at::Tensor &grad,
                                const at::Tensor &grad_per_step,
                                const at::Tensor &cu_seqlens,
                                const std::string &first_half,
                                const std::string &second_half) {
  if (first_half == "add" && second_half == "none") {
    thd_grad_correction_helper<dtype, AddFunctor<dtype>, EmptyFunctor, 0>(
      grad, grad_per_step, cu_seqlens);
  } else if (first_half == "copy" && second_half == "none") {
    thd_grad_correction_helper<dtype, CopyFunctor, EmptyFunctor, 0>(
      grad, grad_per_step, cu_seqlens);
  } else if (first_half == "none" && second_half == "add") {
    thd_grad_correction_helper<dtype, EmptyFunctor, AddFunctor<dtype>, 1>(
      grad, grad_per_step, cu_seqlens);
  } else if (first_half == "none" && second_half == "copy") {
    thd_grad_correction_helper<dtype, EmptyFunctor, CopyFunctor, 1>(
      grad, grad_per_step, cu_seqlens);
  } else if (first_half == "add" && second_half == "copy") {
    thd_grad_correction_helper<dtype, AddFunctor<dtype>, CopyFunctor, 2>(
      grad, grad_per_step, cu_seqlens);
  } else if (first_half == "copy" && second_half == "add") {
    thd_grad_correction_helper<dtype, CopyFunctor, AddFunctor<dtype>, 2>(
      grad, grad_per_step, cu_seqlens);
  } else {
    NVTE_ERROR("Unsupported Functor of first half and second_half\n");
  }
}

void thd_grad_correction(at::Tensor &grad,
                         const at::Tensor &grad_per_step,
                         const at::Tensor &cu_seqlens,
                         const std::string &first_half,
                         const std::string &second_half) {
  if (grad.scalar_type() == at::ScalarType::Half) {
    thd_grad_dispatcher<at::Half>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else if (grad.scalar_type() == at::ScalarType::BFloat16) {
    thd_grad_dispatcher<at::BFloat16>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else if (grad.scalar_type() == at::ScalarType::Float) {
    thd_grad_dispatcher<float>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else {
    NVTE_ERROR("Unsupported dtype of grad\n");
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel:
 * Handle the situation where seqlen is not divisible by cp*2
 **************************************************************************************************/

__global__ void thd_padding_kernel(void *dst,
                                   const void *src,
                                   const int *cu_seqlens_dst,
                                   const int *cu_seqlens_src,
                                   int batch,
                                   int hidden_size_in_bytes,
                                   int dst_len) {
  extern __shared__ int smem[];
  int *smem_dst = &smem[0];
  int *smem_src = &smem[batch + 1];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    smem_dst[i] = cu_seqlens_dst[i];
    smem_src[i] = cu_seqlens_src[i];
  }
  __syncthreads();

  const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / 32;
  const int num_warps = (blockDim.x * gridDim.x) / 32;
  const int laneid = threadIdx.x % 32;

  for (int i = warpid; i < smem_dst[batch]; i += num_warps) {
    float4 *dst_token = (float4*)((char*)dst + i * (size_t)hidden_size_in_bytes);

    int seqid = binary_search(i, smem_dst, batch + 1);
    int offset = i - smem_dst[seqid] + smem_src[seqid];

    if (offset < smem_src[seqid + 1]) {
      float4 *src_token = (float4*)((char*)src + offset * (size_t)hidden_size_in_bytes);
      for (int j = laneid; j < hidden_size_in_bytes / sizeof(float4); j += 32) {
        dst_token[j] = src_token[j];
      }
    } else {
      for (int j = laneid; j < hidden_size_in_bytes / sizeof(float4); j += 32) {
        dst_token[j] = float4{0.0f, 0.0f, 0.0f, 0.0f};
      }
    }
  }

  for (int i = smem_dst[batch] + warpid; i < dst_len; i += num_warps) {
    float4 *dst_token = (float4*)((char*)dst + i * (size_t)hidden_size_in_bytes);
    for (int j = laneid; j < hidden_size_in_bytes / sizeof(float4); j += 32) {
      dst_token[j] = float4{0.0f, 0.0f, 0.0f, 0.0f};
    }
  }
}

at::Tensor thd_padding(const at::Tensor &tensor,
                       const at::Tensor &cu_seqlens,
                       const at::Tensor &cu_seqlens_padded,
                       int output_len,
                       bool reverse) {
  NVTE_CHECK(tensor.dim() == 3);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);
  NVTE_CHECK(cu_seqlens_padded.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens_padded.dim() == 1);
  NVTE_CHECK(cu_seqlens_padded.size(0) == cu_seqlens.size(0));
  if (reverse) {
    NVTE_CHECK(tensor.size(0) >= output_len);
  } else {
    NVTE_CHECK(tensor.size(0) <= output_len);
  }

  int batch        = cu_seqlens_padded.size(0) - 1;
  int num_heads    = tensor.size(1);
  int dim_per_head = tensor.size(2);
  size_t hidden_size_in_bytes = num_heads * dim_per_head * \
                                c10::elementSize(tensor.scalar_type());
  // For 128-bit load/stroe
  NVTE_CHECK(hidden_size_in_bytes % 16 == 0);

  std::vector<int64_t> shape{output_len, num_heads, dim_per_head};
  at::Tensor output_tensor = at::empty(shape, at::CUDA(tensor.scalar_type()));

  const int *cu_seqlens_dst;
  const int *cu_seqlens_src;
  if (reverse) {
    cu_seqlens_dst = cu_seqlens.data_ptr<int>();
    cu_seqlens_src = cu_seqlens_padded.data_ptr<int>();
  } else {
    cu_seqlens_src = cu_seqlens.data_ptr<int>();
    cu_seqlens_dst = cu_seqlens_padded.data_ptr<int>();
  }

  constexpr unsigned int block = 256;
  unsigned int grid = (output_len * 32 + block - 1) / block;
  thd_padding_kernel<<<grid, block, sizeof(int) * (batch+1) * 2,
                       at::cuda::getCurrentCUDAStream()>>>(
    output_tensor.data_ptr(),
    tensor.data_ptr(),
    cu_seqlens_dst,
    cu_seqlens_src,
    batch,
    hidden_size_in_bytes,
    output_len);

  return output_tensor;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Generate partitioned indices for input tokens
 **************************************************************************************************/

__global__ void thd_partition_indices_kernel(int *output,
                                             int *cu_seqlens,
                                             int *cu_seqlens_global,
                                             int batch,
                                             int world_size,
                                             int rank) {
  extern __shared__ int cu_seqlens_s[];
  int *cu_seqlens_g = &cu_seqlens_s[batch+1];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i];
    cu_seqlens_g[i] = cu_seqlens_global[i];
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  int total_tokens = cu_seqlens_s[batch];

  for (int token_id = tid; token_id < total_tokens; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    int seq_len = cu_seqlens_g[seq_id + 1] - cu_seqlens_g[seq_id];
    int n = (seq_len + (2*world_size) - 1) / (2*world_size);
    int index = token_id - cu_seqlens_s[seq_id];
    int offset = index < n ? rank : (2*world_size - 2 - rank);
    index += cu_seqlens_g[seq_id] + n * offset;
    output[token_id] = index;
  }
}

at::Tensor thd_get_partitioned_indices(const at::Tensor &cu_seqlens,
                                       const at::Tensor &cu_seqlens_global,
                                       int output_len,
                                       int world_size,
                                       int rank) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);
  NVTE_CHECK(cu_seqlens_global.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens_global.dim() == 1);
  NVTE_CHECK(cu_seqlens_global.size(0) == cu_seqlens.size(0));
  NVTE_CHECK(output_len > 0);
  NVTE_CHECK(rank >= 0 && rank < world_size);

  int batch = cu_seqlens.size(0) - 1;

  std::vector<int64_t> shape = {output_len};
  at::Tensor output = at::empty(shape, at::CUDA(at::ScalarType::Int));

  constexpr unsigned int block = 256;
  unsigned int grid = (output.size(0) + block - 1) / block;
  thd_partition_indices_kernel<<<grid, block, sizeof(int) * (batch+1) * 2,
                                 at::cuda::getCurrentCUDAStream()>>>(
    output.data_ptr<int>(),
    cu_seqlens.data_ptr<int>(),
    cu_seqlens_global.data_ptr<int>(),
    batch,
    world_size,
    rank);

  return output;
}
